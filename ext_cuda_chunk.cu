#include <cstdio>
#include <cstdarg>
#include "ext_cuda_chunk.hpp"

// Globally shared data structure.
std::vector<TeaLeafCudaChunk*> Chunks;

// Entry point for the initialisation of the CUDA extension
extern "C"
void ext_init_cuda_(
		int* xMax, 
		int* yMax, 
		int* rank)
{
	Chunks.push_back(new TeaLeafCudaChunk(*xMax, *yMax, *rank));
}

// Entry point for the finalisation of the CUDA extension
extern "C"
void ext_finalise_()
{
	for(int ii = 0; ii != Chunks.size(); ++ii)
	{
		delete Chunks[ii];
	}
}

TeaLeafCudaChunk::TeaLeafCudaChunk(
		int xMax, 
		int yMax, 
		int rank)
: xCells(xMax+HALO_PAD*2), 
	yCells(yMax+HALO_PAD*2),
	rank(rank)
{
	// Naive assumption that devices are paired even and odd
	int numDevices;
	hipGetDeviceCount(&numDevices);
	deviceId = rank%numDevices;

	int result = hipSetDevice(deviceId);
	if(result != hipSuccess)
	{
		Abort(__LINE__,__FILE__,"Could not allocate CUDA device %d.\n", deviceId);
	}

	struct hipDeviceProp_t properties;
	hipGetDeviceProperties(&properties, deviceId);

	printf("Rank %d using %s device id %d\n", rank, properties.name, deviceId);

	const int block = xCells*yCells;

#define CUDA_MALLOC(buf, size) 		\
	hipMalloc((void**)&buf, size);	\
	CheckErrors(__LINE__,__FILE__);	\
	hipDeviceSynchronize();		\
	hipMemset(buf, 0, size);		\
	hipDeviceSynchronize();		\
	CheckErrors(__LINE__,__FILE__);

	CUDA_MALLOC(dU, block*sizeof(double));
	CUDA_MALLOC(dU0, block*sizeof(double));
	CUDA_MALLOC(dSd, block*sizeof(double));
	CUDA_MALLOC(dR, block*sizeof(double));
	CUDA_MALLOC(dW, block*sizeof(double));
	CUDA_MALLOC(dZ, block*sizeof(double));
	CUDA_MALLOC(dP, block*sizeof(double));
	CUDA_MALLOC(dMi, block*sizeof(double));
	CUDA_MALLOC(dD, block*sizeof(double));
	CUDA_MALLOC(dKx, block*sizeof(double));
	CUDA_MALLOC(dKy, block*sizeof(double));
	CUDA_MALLOC(dDensity, block*sizeof(double));
	CUDA_MALLOC(dEnergy0, block*sizeof(double));
	CUDA_MALLOC(dEnergy1, block*sizeof(double));
	CUDA_MALLOC(dVolume, block*sizeof(double));
	CUDA_MALLOC(dCellX, xCells*sizeof(double));
	CUDA_MALLOC(dCellY, yCells*sizeof(double));
	CUDA_MALLOC(dCellDx, xCells*sizeof(double));
	CUDA_MALLOC(dCellDy, yCells*sizeof(double));
	CUDA_MALLOC(dVertexX, (xCells+1)*sizeof(double));
	CUDA_MALLOC(dVertexY, (yCells+1)*sizeof(double));
	CUDA_MALLOC(dVertexDx, (xCells+1)*sizeof(double));
	CUDA_MALLOC(dVertexDy, (yCells+1)*sizeof(double));
	CUDA_MALLOC(dXArea, (xCells+1)*yCells*sizeof(double));
	CUDA_MALLOC(dYArea, xCells*(yCells+1)*sizeof(double));
	CUDA_MALLOC(dTopBuffer, (xCells+1)*MAX_DEPTH*NUM_FIELDS*sizeof(double));
	CUDA_MALLOC(dBottomBuffer, (xCells+1)*MAX_DEPTH*NUM_FIELDS*sizeof(double));
	CUDA_MALLOC(dLeftBuffer, (yCells+1)*MAX_DEPTH*NUM_FIELDS*sizeof(double));
	CUDA_MALLOC(dRightBuffer, (yCells+1)*MAX_DEPTH*NUM_FIELDS*sizeof(double));

	const size_t reduceBufferLength = xCells*yCells*sizeof(double);
	CUDA_MALLOC(dReduceBuffer1, reduceBufferLength);
	CUDA_MALLOC(dReduceBuffer2, reduceBufferLength);
	CUDA_MALLOC(dReduceBuffer3, reduceBufferLength);
	CUDA_MALLOC(dReduceBuffer4, reduceBufferLength);
#undef CUDA_MALLOC
}

TeaLeafCudaChunk::~TeaLeafCudaChunk()
{
#ifdef ENABLE_PROFILING
	double totalTime = 0.0;

	fprintf(stdout, "%30s %7s %5s %9s\n", "Kernel name", "runtime", "calls", "bandwidth");
	std::map<std::string, double>::iterator ii = kernelTimes.begin();
	std::map<std::string, int>::iterator jj = kernelCalls.begin();

	for( ; ii != kernelTimes.end(); ++ii, ++jj)
	{
		totalTime += ii->second;

		fprintf(stdout, "%30s %9.3f %5d %7.5f\n", 
				ii->first.c_str(), ii->second, jj->second, 0.0);
	}

	fprintf(stdout, "Total kernel time %f ms\n", totalTime);
#endif
}

// Synchronises and checks for most recent CUDA error.
void TeaLeafCudaChunk::CheckErrors(int lineNum, const char* file)
{
	hipDeviceSynchronize();
	int result = hipGetLastError();

	if(result != hipSuccess)
	{
		Abort(lineNum, file, "Error in %s - return code %d (%s)\n", file, result, CudaCodes(result));
	}
}

// Aborts the application.
void TeaLeafCudaChunk::Abort(int lineNum, const char* file, const char* format, ...)
{
	fprintf(stderr, "\x1b[31m");
	fprintf(stderr, "\nError at line %d in %s:", lineNum, file);
	fprintf(stderr, "\x1b[0m \n");

	va_list arglist;
	va_start(arglist, format);
	vfprintf(stderr, format, arglist);
	va_end(arglist);

	exit(1);
}

// Enumeration for the set of potential CUDA error codes.
const char* TeaLeafCudaChunk::CudaCodes(int code)
{
	switch(code)
	{
		case hipSuccess: return "hipSuccess"; // 0
		case hipErrorMissingConfiguration: return "hipErrorMissingConfiguration"; // 1
		case hipErrorOutOfMemory: return "hipErrorOutOfMemory"; // 2
		case hipErrorNotInitialized: return "hipErrorNotInitialized"; // 3
		case hipErrorLaunchFailure: return "hipErrorLaunchFailure"; // 4
		case hipErrorPriorLaunchFailure: return "hipErrorPriorLaunchFailure"; // 5
		case hipErrorLaunchTimeOut: return "hipErrorLaunchTimeOut"; // 6
		case hipErrorLaunchOutOfResources: return "hipErrorLaunchOutOfResources"; // 7
		case hipErrorInvalidDeviceFunction: return "hipErrorInvalidDeviceFunction"; // 8
		case hipErrorInvalidConfiguration: return "hipErrorInvalidConfiguration"; // 9
		case hipErrorInvalidDevice: return "hipErrorInvalidDevice"; // 10
		case hipErrorInvalidValue: return "hipErrorInvalidValue";// 11
		case hipErrorInvalidPitchValue: return "hipErrorInvalidPitchValue";// 12
		case hipErrorInvalidSymbol: return "hipErrorInvalidSymbol";// 13
		case hipErrorMapFailed: return "hipErrorMapFailed";// 14
		case hipErrorUnmapFailed: return "hipErrorUnmapFailed";// 15
		case cudaErrorInvalidHostPointer: return "cudaErrorInvalidHostPointer";// 16
		case hipErrorInvalidDevicePointer: return "hipErrorInvalidDevicePointer";// 17
		case hipErrorInvalidTexture: return "hipErrorInvalidTexture";// 18
		case cudaErrorInvalidTextureBinding: return "cudaErrorInvalidTextureBinding";// 19
		case hipErrorInvalidChannelDescriptor: return "hipErrorInvalidChannelDescriptor";// 20
		case hipErrorInvalidMemcpyDirection: return "hipErrorInvalidMemcpyDirection";// 21
		case cudaErrorAddressOfConstant: return "cudaErrorAddressOfConstant";// 22
		case cudaErrorTextureFetchFailed: return "cudaErrorTextureFetchFailed";// 23
		case cudaErrorTextureNotBound: return "cudaErrorTextureNotBound";// 24
		case cudaErrorSynchronizationError: return "cudaErrorSynchronizationError";// 25
		case cudaErrorInvalidFilterSetting: return "cudaErrorInvalidFilterSetting";// 26
		case cudaErrorInvalidNormSetting: return "cudaErrorInvalidNormSetting";// 27
		case cudaErrorMixedDeviceExecution: return "cudaErrorMixedDeviceExecution";// 28
		case hipErrorDeinitialized: return "hipErrorDeinitialized";// 29
		case hipErrorUnknown: return "hipErrorUnknown";// 30
		case cudaErrorNotYetImplemented: return "cudaErrorNotYetImplemented";// 31
		case cudaErrorMemoryValueTooLarge: return "cudaErrorMemoryValueTooLarge";// 32
		case hipErrorInvalidHandle: return "hipErrorInvalidHandle";// 33
		case hipErrorNotReady: return "hipErrorNotReady";// 34
		case hipErrorInsufficientDriver: return "hipErrorInsufficientDriver";// 35
		case hipErrorSetOnActiveProcess: return "hipErrorSetOnActiveProcess";// 36
		case cudaErrorInvalidSurface: return "cudaErrorInvalidSurface";// 37
		case hipErrorNoDevice: return "hipErrorNoDevice";// 38
		case hipErrorECCNotCorrectable: return "hipErrorECCNotCorrectable";// 39
		case hipErrorSharedObjectSymbolNotFound: return "hipErrorSharedObjectSymbolNotFound";// 40
		case hipErrorSharedObjectInitFailed: return "hipErrorSharedObjectInitFailed";// 41
		case hipErrorUnsupportedLimit: return "hipErrorUnsupportedLimit";// 42
		case cudaErrorDuplicateVariableName: return "cudaErrorDuplicateVariableName";// 43
		case cudaErrorDuplicateTextureName: return "cudaErrorDuplicateTextureName";// 44
		case cudaErrorDuplicateSurfaceName: return "cudaErrorDuplicateSurfaceName";// 45
		case cudaErrorDevicesUnavailable: return "cudaErrorDevicesUnavailable";// 46
		case hipErrorInvalidImage: return "hipErrorInvalidImage";// 47
		case hipErrorNoBinaryForGpu: return "hipErrorNoBinaryForGpu";// 48
		case cudaErrorIncompatibleDriverContext: return "cudaErrorIncompatibleDriverContext";// 49
		case hipErrorPeerAccessAlreadyEnabled: return "hipErrorPeerAccessAlreadyEnabled";// 50
		case hipErrorPeerAccessNotEnabled: return "hipErrorPeerAccessNotEnabled";// 51
		case hipErrorContextAlreadyInUse: return "hipErrorContextAlreadyInUse";// 52
		case hipErrorProfilerDisabled: return "hipErrorProfilerDisabled";// 53
		case hipErrorProfilerNotInitialized: return "hipErrorProfilerNotInitialized";// 54
		case hipErrorProfilerAlreadyStarted: return "hipErrorProfilerAlreadyStarted";// 55
		case hipErrorProfilerAlreadyStopped: return "hipErrorProfilerAlreadyStopped";// 56
		case hipErrorAssert: return "hipErrorAssert";// 57
		case cudaErrorTooManyPeers: return "cudaErrorTooManyPeers";// 58
		case hipErrorHostMemoryAlreadyRegistered: return "hipErrorHostMemoryAlreadyRegistered";// 59
		case hipErrorHostMemoryNotRegistered: return "hipErrorHostMemoryNotRegistered";// 60
		case hipErrorOperatingSystem: return "hipErrorOperatingSystem";// 61
		case cudaErrorStartupFailure: return "cudaErrorStartupFailure";// 62
		case cudaErrorApiFailureBase: return "cudaErrorApiFailureBase";// 63
		default: return "Unknown error";
	}
}

