#include "hip/hip_runtime.h"
#include <cstdio>
#include <numeric>
#include <iostream>
#include "ext_cuda_chunk.hpp"
#include "kernels/pack_kernel.cuknl"

using std::ceil;
using std::accumulate;

#define CELL_DATA 1
#define VERTEX_DATA 2
#define X_FACE_DATA 3
#define Y_FACE_DATA 4
#define WARP_SIZE 32.0

// Extended kernel for packing messages
extern "C"
void ext_pack_message_(
		const int* chunk,
		const int* fields,
		const int* offsets,
		const int* depth,
		const int* face,
		double* buffer)
{
	Chunks[*chunk-1]->PackUnpackKernel(fields, offsets, *depth, *face, buffer, true);
}

// Extended kernel for unpacking messages
extern "C"
void ext_unpack_message_(
		const int* chunk,
		const int* fields,
		const int* offsets,
		const int* depth,
		const int* face,
		double* buffer)
{
	Chunks[*chunk-1]->PackUnpackKernel(fields, offsets, *depth, *face, buffer, false);
}

// Packs or unpacks communication buffers
void TeaLeafCudaChunk::PackUnpackKernel(
		const int* fields,
		const int* offsets,
		const int depth,
		const int face,
		double* buffer,
		const bool pack)
{
	const int exchanges = accumulate(fields, fields+NUM_FIELDS, 0);

	if(exchanges < 1) return;

	std::string kernelName;
	double* deviceBuffer = NULL;
	CuKnlPackType packKernel = NULL;

	int bufferLength = 0;
	int innerX = xCells-2*HALO_PAD;
	int innerY = yCells-2*HALO_PAD;
	
	switch(face)
	{
		case CHUNK_LEFT:
			kernelName = (pack) ? "Pack Left" : "Unpack Left";
			packKernel = (pack) ? CuKnlPackLeft : CuKnlUnpackLeft;
			deviceBuffer = dLeftBuffer;
			bufferLength = innerY*depth;
			break;
		case CHUNK_RIGHT:
			kernelName = (pack) ? "Pack Right" : "Unpack Right";
			packKernel = (pack) ? CuKnlPackRight : CuKnlUnpackRight;
			deviceBuffer = dRightBuffer;
			bufferLength = innerY*depth;
			break;
		case CHUNK_TOP:
			kernelName = (pack) ? "Pack Top" : "Unpack Top";
			packKernel = (pack) ? CuKnlPackTop : CuKnlUnpackTop;
			deviceBuffer = dTopBuffer;
			bufferLength = innerX*depth;
			break;
		case CHUNK_BOTTOM:
			kernelName = (pack) ? "Pack Bottom" : "Unpack Bottom";
			packKernel = (pack) ? CuKnlPackBottom : CuKnlUnpackBottom;
			deviceBuffer = dBottomBuffer;
			bufferLength = innerX*depth;
			break;
		default:
			TeaLeafCudaChunk::Abort(__LINE__, __FILE__, 
					"Incorrect face provided: %d.\n", face);
	}

	if(!pack)
	{
		hipMemcpy(deviceBuffer, buffer, exchanges*bufferLength*sizeof(double), 
				hipMemcpyHostToDevice);
		TeaLeafCudaChunk::CheckErrors(__LINE__,__FILE__);
	}

	int offset = 0;
	int numBlocks = ceil(bufferLength/(float)BLOCK_SIZE);

	for(int ii = 0; ii != NUM_FIELDS; ++ii)
	{
		if(fields[ii])
		{
			double* deviceField = NULL;
			switch(ii+1)
			{
				case FIELD_DENSITY:
					deviceField = dDensity;
					break;
				case FIELD_ENERGY0:
					deviceField = dEnergy0;
					break;
				case FIELD_ENERGY1:
					deviceField = dEnergy1;
					break;
				case FIELD_U:
					deviceField = dU;
					break;
				case FIELD_P:
					deviceField = dP;
					break;
				case FIELD_SD:
					deviceField = dSd;
					break;
				default:
					TeaLeafCudaChunk::Abort(__LINE__,__FILE__,
							"Incorrect field provided: %d.\n", ii+1);
			}

			START_PROFILING();

			int bufferOffset = bufferLength*offset++;
			packKernel<<<numBlocks, BLOCK_SIZE>>>(
					xCells, yCells, innerX, innerY, 
					deviceField, deviceBuffer+bufferOffset, depth);

			POST_KERNEL(kernelName.c_str());
		}
	}

	if(pack)
	{
		hipMemcpy(buffer, deviceBuffer, exchanges*bufferLength*sizeof(double),
				hipMemcpyDeviceToHost);
		TeaLeafCudaChunk::CheckErrors(__LINE__,__FILE__);
	}
}
