#include "hip/hip_runtime.h"
/*Crown Copyright 2012 AWE.
 *
 * This file is part of CloverLeaf.
 *
 * CloverLeaf is free software: you can redistribute it and/or modify it under
 * the terms of the GNU General Public License as published by the
 * Free Software Foundation, either version 3 of the License, or (at your option)
 * any later version.
 *
 * CloverLeaf is distributed in the hope that it will be useful, but
 * WITHOUT ANY WARRANTY; without even the implied warranty of MERCHANTABILITY or
 * FITNESS FOR A PARTICULAR PURPOSE. See the GNU General Public License for more
 * details.
 *
 * You should have received a copy of the GNU General Public License along with
 * CloverLeaf. If not, see http://www.gnu.org/licenses/.
 */

/*
 *  @brief CUDA kernel to update the external halo cells in a chunk.
 *  @author Michael Boulton NVIDIA Corporation
 *  @details Updates halo cells for the required fields at the required depth
 *  for any halo cells that lie on an external boundary. The location and type
 *  of data governs how this is carried out. External boundaries are always
 *  reflective.
 */

#include "cuda_common.hpp"
#include "kernel_files/update_halo_kernel.cuknl"

extern "C" void update_halo_kernel_cuda_
(const int* chunk_neighbours,
const int* fields,
const int* depth)
{
    cuda_chunk.update_halo_kernel(fields, *depth, chunk_neighbours);
}

void update_array
(int x_min, int x_max, int y_min, int y_max,
cell_info_t const& grid_type,
const int* chunk_neighbours,
double* cur_array_d,
int depth)
{
    #define CHECK_LAUNCH(face, dir)                                 \
    if (EXTERNAL_FACE == chunk_neighbours[CHUNK_ ## face])          \
    {                                                               \
        const int launch_sz = (ceil((dir##_max+4+grid_type.dir##_extra) \
            /static_cast<float>(BLOCK_SZ))) * depth;                \
        device_update_halo_kernel_##face##_cuda                     \
        <<< launch_sz, BLOCK_SZ >>>                                 \
        (x_min, x_max, y_min, y_max, grid_type, cur_array_d, depth);\
    }

    CHECK_LAUNCH(bottom, x);
    CHECK_LAUNCH(top, x);
    CHECK_LAUNCH(left, y);
    CHECK_LAUNCH(right, y);

    #undef CHECK_LAUNCH
}

void CloverleafCudaChunk::update_halo_kernel
(const int* fields,
const int depth,
const int* chunk_neighbours)
{
    CUDA_BEGIN_PROFILE;

    #define HALO_UPDATE_RESIDENT(arr, grid_type)        \
    {if (1 == fields[FIELD_##arr - 1])                  \
    {                                                   \
        update_array(x_min, x_max, y_min, y_max,        \
            grid_type, chunk_neighbours, arr, depth);   \
        CUDA_ERR_CHECK;                                 \
    }}

    HALO_UPDATE_RESIDENT(density0, CELL);
    HALO_UPDATE_RESIDENT(density1, CELL);
    HALO_UPDATE_RESIDENT(energy0, CELL);
    HALO_UPDATE_RESIDENT(energy1, CELL);
    HALO_UPDATE_RESIDENT(pressure, CELL);
    HALO_UPDATE_RESIDENT(viscosity, CELL);

    HALO_UPDATE_RESIDENT(xvel0, VERTEX_X);
    HALO_UPDATE_RESIDENT(xvel1, VERTEX_X);

    HALO_UPDATE_RESIDENT(yvel0, VERTEX_Y);
    HALO_UPDATE_RESIDENT(yvel1, VERTEX_Y);

    HALO_UPDATE_RESIDENT(vol_flux_x, X_FACE);
    HALO_UPDATE_RESIDENT(mass_flux_x, X_FACE);

    HALO_UPDATE_RESIDENT(vol_flux_y, Y_FACE);
    HALO_UPDATE_RESIDENT(mass_flux_y, Y_FACE);

    HALO_UPDATE_RESIDENT(u, CELL);
    HALO_UPDATE_RESIDENT(work_array_1, CELL);

    #undef HALO_UPDATE_RESIDENT

    CUDA_END_PROFILE;
}

