#include "hip/hip_runtime.h"
/*Crown Copyright 2012 AWE.
 *
 * This file is part of CloverLeaf.
 *
 * CloverLeaf is free software: you can redistribute it and/or modify it under
 * the terms of the GNU General Public License as published by the
 * Free Software Foundation, either version 3 of the License, or (at your option)
 * any later version.
 *
 * CloverLeaf is distributed in the hope that it will be useful, but
 * WITHOUT ANY WARRANTY; without even the implied warranty of MERCHANTABILITY or
 * FITNESS FOR A PARTICULAR PURPOSE. See the GNU General Public License for more
 * details.
 *
 * You should have received a copy of the GNU General Public License along with
 * CloverLeaf. If not, see http://www.gnu.org/licenses/.
 */

/*
 *  @brief CUDA mpi buffer transfer
 *  @author Michael Boulton
 *  @details Transfers the buffers required for the mpi halo exchange
 */

#include "cuda_common.hpp"
#include "kernel_files/pack_kernel.cuknl"

#include <numeric>

/**********************/

// define a generic interface for fortran
#define C_PACK_INTERFACE(operation, dir)                            \
extern "C" void operation##_##dir##_buffers_cuda_                    \
(int *xmin, int *xmax, int *ymin, int *ymax,                        \
 int *chunk_1, int *chunk_2, int *external_face,                    \
 int *x_inc, int *y_inc, int *depth, int *which_field,              \
 double *field_ptr, double *buffer_1, double *buffer_2)             \
{                                                                   \
    cuda_chunk.operation##_##dir(*chunk_1, *chunk_2, *external_face,     \
                            *x_inc, *y_inc, *depth,                 \
                            (*which_field)-1, buffer_1, buffer_2);  \
}

C_PACK_INTERFACE(pack, left_right)
C_PACK_INTERFACE(unpack, left_right)
C_PACK_INTERFACE(pack, top_bottom)
C_PACK_INTERFACE(unpack, top_bottom)

/*****************************/

void CloverleafCudaChunk::packRect
(double* host_buffer, dir_t direction,
 int x_inc, int y_inc, int edge, int dest,
 int which_field, int depth)
{
    // TODO just call packBuffer/unpackBuffer from this
    // TODO remove unnecesary synchronisations below
}

void CloverleafCudaChunk::packBuffer
(const int which_array,
const int which_side,
double* buffer,
const int buffer_size,
const int depth)
{
    #define CALL_PACK(dev_ptr, type, face, dir)\
	{\
        const int launch_sz = (ceil((dir##_max+4+type.dir##_extra)/static_cast<float>(BLOCK_SZ))) * depth; \
        device_pack##face##Buffer<<< launch_sz, BLOCK_SZ >>> \
        (x_min, x_max, y_min, y_max, type, \
        dev_ptr, dev_##face##_send_buffer, depth); \
        CUDA_ERR_CHECK; \
        hipMemcpy(buffer, dev_##face##_send_buffer, buffer_size*sizeof(double), hipMemcpyDeviceToHost); \
        CUDA_ERR_CHECK; \
        hipDeviceSynchronize();\
        break; \
	}

    #define PACK_CUDA_BUFFERS(dev_ptr, type) \
        switch(which_side) \
        { \
            case CHUNK_LEFT: \
                CALL_PACK(dev_ptr, type, left, y);\
            case CHUNK_RIGHT:\
                CALL_PACK(dev_ptr, type, right, y);\
            case CHUNK_BOTTOM:\
                CALL_PACK(dev_ptr, type, bottom, x);\
            case CHUNK_TOP:\
                CALL_PACK(dev_ptr, type, top, x);\
            default: \
                DIE("Invalid side passed to buffer packing"); \
        }

    switch(which_array)
    {
        case FIELD_density0: PACK_CUDA_BUFFERS(density0, CELL); break;
        case FIELD_density1: PACK_CUDA_BUFFERS(density1, CELL); break;
        case FIELD_energy0: PACK_CUDA_BUFFERS(energy0, CELL); break;
        case FIELD_energy1: PACK_CUDA_BUFFERS(energy1, CELL); break;
        case FIELD_pressure: PACK_CUDA_BUFFERS(pressure, CELL); break;
        case FIELD_viscosity: PACK_CUDA_BUFFERS(viscosity, CELL); break;
        case FIELD_soundspeed: PACK_CUDA_BUFFERS(soundspeed, CELL); break;
        case FIELD_xvel0: PACK_CUDA_BUFFERS(xvel0, VERTEX_X); break;
        case FIELD_xvel1: PACK_CUDA_BUFFERS(xvel1, VERTEX_X); break;
        case FIELD_yvel0: PACK_CUDA_BUFFERS(yvel0, VERTEX_Y); break;
        case FIELD_yvel1: PACK_CUDA_BUFFERS(yvel1, VERTEX_Y); break;
        case FIELD_vol_flux_x: PACK_CUDA_BUFFERS(vol_flux_x, X_FACE); break;
        case FIELD_vol_flux_y: PACK_CUDA_BUFFERS(vol_flux_y, Y_FACE); break;
        case FIELD_mass_flux_x: PACK_CUDA_BUFFERS(mass_flux_x, X_FACE); break;
        case FIELD_mass_flux_y: PACK_CUDA_BUFFERS(mass_flux_y, Y_FACE); break;
        default: DIE("Invalid which_array identifier passed to CUDA");
    }

}

void CloverleafCudaChunk::unpackBuffer
(const int which_array,
const int which_side,
double* buffer,
const int buffer_size,
const int depth)
{
    #define CALL_UNPACK(dev_ptr, type, face, dir)\
	{ \
        hipMemcpy(dev_##face##_recv_buffer, buffer, buffer_size*sizeof(double), hipMemcpyHostToDevice); \
        CUDA_ERR_CHECK; \
        hipDeviceSynchronize();\
        const int launch_sz = (ceil((dir##_max+4+type.dir##_extra)/static_cast<float>(BLOCK_SZ))) * depth; \
        device_unpack##face##Buffer<<< launch_sz, BLOCK_SZ >>> \
        (x_min, x_max, y_min, y_max, type, \
        dev_ptr, dev_##face##_recv_buffer, depth); \
        CUDA_ERR_CHECK; \
        break; \
	}

    #define UNPACK_CUDA_BUFFERS(dev_ptr, type) \
        switch(which_side) \
        { \
            case CHUNK_LEFT: \
                CALL_UNPACK(dev_ptr, type, left, y);\
            case CHUNK_RIGHT:\
                CALL_UNPACK(dev_ptr, type, right, y);\
            case CHUNK_BOTTOM:\
                CALL_UNPACK(dev_ptr, type, bottom, x);\
            case CHUNK_TOP:\
                CALL_UNPACK(dev_ptr, type, top, x);\
            default: \
                DIE("Invalid side passed to buffer unpacking"); \
        }

    switch(which_array)
    {
        case FIELD_density0: UNPACK_CUDA_BUFFERS(density0, CELL); break;
        case FIELD_density1: UNPACK_CUDA_BUFFERS(density1, CELL); break;
        case FIELD_energy0: UNPACK_CUDA_BUFFERS(energy0, CELL); break;
        case FIELD_energy1: UNPACK_CUDA_BUFFERS(energy1, CELL); break;
        case FIELD_pressure: UNPACK_CUDA_BUFFERS(pressure, CELL); break;
        case FIELD_viscosity: UNPACK_CUDA_BUFFERS(viscosity, CELL); break;
        case FIELD_soundspeed: UNPACK_CUDA_BUFFERS(soundspeed, CELL); break;
        case FIELD_xvel0: UNPACK_CUDA_BUFFERS(xvel0, VERTEX_X); break;
        case FIELD_xvel1: UNPACK_CUDA_BUFFERS(xvel1, VERTEX_X); break;
        case FIELD_yvel0: UNPACK_CUDA_BUFFERS(yvel0, VERTEX_Y); break;
        case FIELD_yvel1: UNPACK_CUDA_BUFFERS(yvel1, VERTEX_Y); break;
        case FIELD_vol_flux_x: UNPACK_CUDA_BUFFERS(vol_flux_x, X_FACE); break;
        case FIELD_vol_flux_y: UNPACK_CUDA_BUFFERS(vol_flux_y, Y_FACE); break;
        case FIELD_mass_flux_x: UNPACK_CUDA_BUFFERS(mass_flux_x, X_FACE); break;
        case FIELD_mass_flux_y: UNPACK_CUDA_BUFFERS(mass_flux_y, Y_FACE); break;
        default: DIE("Invalid which_array identifier passed to CUDA");
    }
}

int CloverleafCudaChunk::getBufferSize
(int edge, int depth, int x_inc, int y_inc)
{
    int region[2];

    switch (edge)
    {
    // depth*y_max+... region - 1 or 2 columns
    case CHUNK_LEFT:
        region[0] = depth;
        region[1] = y_max + y_inc + (2*depth);
        break;
    case CHUNK_RIGHT:
        region[0] = depth;
        region[1] = y_max + y_inc + (2*depth);
        break;

    // depth*x_max+... region - 1 or 2 rows
    case CHUNK_BOTTOM:
        region[0] = x_max + x_inc + (2*depth);
        region[1] = depth;
        break;
    case CHUNK_TOP:
        region[0] = x_max + x_inc + (2*depth);
        region[1] = depth;
        break;
    default:
        DIE("Invalid face identifier (%d) passed to getBufferSize\n");
    }

    return region[0]*region[1];
}

#define CHECK_PACK(op, side1, side2)                          \
    if (external_face != chunk_1 || external_face != chunk_2)               \
    {                                                                       \
        hipDeviceSynchronize();                                            \
    } \
    if (external_face != chunk_1)                                           \
    {                                                                       \
        op##Buffer(which_field, \
                   chunk_1, \
                   buffer_1, \
                   getBufferSize(chunk_1, depth, x_inc, y_inc), \
                   depth); \
    }                                                                       \
    if (external_face != chunk_2)                                           \
    {                                                                       \
        op##Buffer(which_field, \
                   chunk_2, \
                   buffer_2, \
                   getBufferSize(chunk_2, depth, x_inc, y_inc), \
                   depth); \
    }                                                                       \
    if (external_face != chunk_1 || external_face != chunk_2)               \
    {                                                                       \
        hipDeviceSynchronize();                                            \
    }

void CloverleafCudaChunk::pack_left_right
(PACK_ARGS)
{
    CHECK_PACK(pack, CHUNK_LEFT, CHUNK_RIGHT);
}

void CloverleafCudaChunk::unpack_left_right
(PACK_ARGS)
{
    CHECK_PACK(unpack, CHUNK_LEFT, CHUNK_RIGHT);
}

void CloverleafCudaChunk::pack_top_bottom
(PACK_ARGS)
{
    CHECK_PACK(pack, CHUNK_BOTTOM, CHUNK_TOP);
}

void CloverleafCudaChunk::unpack_top_bottom
(PACK_ARGS)
{
    CHECK_PACK(unpack, CHUNK_BOTTOM, CHUNK_TOP);
}

