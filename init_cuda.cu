/*Crown Copyright 2012 AWE.
 *
 * This file is part of CloverLeaf.
 *
 * CloverLeaf is free software: you can redistribute it and/or modify it under
 * the terms of the GNU General Public License as published by the
 * Free Software Foundation, either version 3 of the License, or (at your option)
 * any later version.
 *
 * CloverLeaf is distributed in the hope that it will be useful, but
 * WITHOUT ANY WARRANTY; without even the implied warranty of MERCHANTABILITY or
 * FITNESS FOR A PARTICULAR PURPOSE. See the GNU General Public License for more
 * details.
 *
 * You should have received a copy of the GNU General Public License along with
 * CloverLeaf. If not, see http://www.gnu.org/licenses/.
 */

/*
 *  @brief CUDA initialisation
 *  @author Michael Boulton NVIDIA Corporation
 *  @details Initialises CUDA devices and global storage
 */

#include "cuda_common.hpp"

#include <sstream>
#include <cstdio>
#include <cassert>

CloverleafCudaChunk cuda_chunk;

extern "C" void initialise_cuda_
(INITIALISE_ARGS)
{
    cuda_chunk = CloverleafCudaChunk(in_x_min,
                                in_x_max,
                                in_y_min,
                                in_y_max,
                                in_profiler_on);
}

CloverleafCudaChunk::CloverleafCudaChunk
(void)
{
    ;
}

static std::string matchParam
(FILE * input,
 const char* param_name)
{
    std::string param_string;
    static char name_buf[101];
    rewind(input);
    /* read in line from file */
    while (NULL != fgets(name_buf, 100, input))
    {
        /* if it has the parameter name, its the line we want */
        if (NULL != strstr(name_buf, param_name))
        {
            if (NULL != strstr(name_buf, "="))
            {
                *(strstr(name_buf, "=")) = ' ';
                char param_buf[100];
                sscanf(name_buf, "%*s %s", param_buf);
                param_string = std::string(param_buf);
                break;
            }
            else
            {
                param_string = std::string("NO_SETTING");
                break;
            }
        }
    }

    return param_string;
}

int preferredDevice
(void)
{
    FILE* input;
    assert(input = fopen("tea.in", "r"));

    std::string param_string = matchParam(input, "cuda_device");

    int preferred_device;

    if (param_string.size() == 0)
    {
        // not found in file
        preferred_device = 0;
        std::cout << "CUDA device not specifiefd in file - using 0" << std::endl;
    }
    else
    {
        std::stringstream converter(param_string);

        if (!(converter >> preferred_device))
        {
            preferred_device = -1;
        }
    }

    fclose(input);

    return preferred_device;
}

CloverleafCudaChunk::CloverleafCudaChunk
(INITIALISE_ARGS)
:x_min(*in_x_min),
x_max(*in_x_max),
y_min(*in_y_min),
y_max(*in_y_max),
profiler_on(*in_profiler_on),
num_blocks((((*in_x_max)+5)*((*in_y_max)+5))/BLOCK_SZ)
{
    // FIXME (and opencl really)
    // make a better platform agnostic way of selecting devices

    // choose device 0 unless specified
    hipDeviceReset();
    int device_id = preferredDevice();
    hipSetDevice(device_id); 

    struct hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, device_id);
    std::cout << "CUDA using " << prop.name << std::endl;

    #define CUDA_ARRAY_ALLOC(arr, size)     \
            hipMalloc((void**) &arr, size);\
            hipDeviceSynchronize();        \
            hipMemset(arr, 0, size);       \
            hipDeviceSynchronize();        \
            CUDA_ERR_CHECK;

    CUDA_ARRAY_ALLOC(volume, BUFSZ2D(0, 0));
    CUDA_ARRAY_ALLOC(soundspeed, BUFSZ2D(0, 0));
    CUDA_ARRAY_ALLOC(pressure, BUFSZ2D(0, 0));
    CUDA_ARRAY_ALLOC(viscosity, BUFSZ2D(0, 0));

    CUDA_ARRAY_ALLOC(density0, BUFSZ2D(0, 0));
    CUDA_ARRAY_ALLOC(density1, BUFSZ2D(0, 0));
    CUDA_ARRAY_ALLOC(energy0, BUFSZ2D(0, 0));
    CUDA_ARRAY_ALLOC(energy1, BUFSZ2D(0, 0));

    CUDA_ARRAY_ALLOC(u, BUFSZ2D(0, 0));
    CUDA_ARRAY_ALLOC(u0, BUFSZ2D(0, 0));

    CUDA_ARRAY_ALLOC(xvel0, BUFSZ2D(1, 1));
    CUDA_ARRAY_ALLOC(xvel1, BUFSZ2D(1, 1));
    CUDA_ARRAY_ALLOC(yvel0, BUFSZ2D(1, 1));
    CUDA_ARRAY_ALLOC(yvel1, BUFSZ2D(1, 1));

    CUDA_ARRAY_ALLOC(xarea, BUFSZ2D(1, 0));
    CUDA_ARRAY_ALLOC(vol_flux_x, BUFSZ2D(1, 0));
    CUDA_ARRAY_ALLOC(mass_flux_x, BUFSZ2D(1, 0));

    CUDA_ARRAY_ALLOC(yarea, BUFSZ2D(0, 1));
    CUDA_ARRAY_ALLOC(vol_flux_y, BUFSZ2D(0, 1));
    CUDA_ARRAY_ALLOC(mass_flux_y, BUFSZ2D(0, 1));

    CUDA_ARRAY_ALLOC(cellx, BUFSZX(0));
    CUDA_ARRAY_ALLOC(celldx, BUFSZX(0));
    CUDA_ARRAY_ALLOC(vertexx, BUFSZX(1));
    CUDA_ARRAY_ALLOC(vertexdx, BUFSZX(1));

    CUDA_ARRAY_ALLOC(celly, BUFSZY(0));
    CUDA_ARRAY_ALLOC(celldy, BUFSZY(0));
    CUDA_ARRAY_ALLOC(vertexy, BUFSZY(1));
    CUDA_ARRAY_ALLOC(vertexdy, BUFSZY(1));

    CUDA_ARRAY_ALLOC(work_array_1, BUFSZ2D(1, 1));
    CUDA_ARRAY_ALLOC(work_array_2, BUFSZ2D(1, 1));
    CUDA_ARRAY_ALLOC(work_array_3, BUFSZ2D(1, 1));
    CUDA_ARRAY_ALLOC(work_array_4, BUFSZ2D(1, 1));
    CUDA_ARRAY_ALLOC(work_array_5, BUFSZ2D(1, 1));
    CUDA_ARRAY_ALLOC(work_array_6, BUFSZ2D(1, 1));

    CUDA_ARRAY_ALLOC(reduce_buf_1, num_blocks*sizeof(double));
    CUDA_ARRAY_ALLOC(reduce_buf_2, num_blocks*sizeof(double));
    CUDA_ARRAY_ALLOC(reduce_buf_3, num_blocks*sizeof(double));
    CUDA_ARRAY_ALLOC(reduce_buf_4, num_blocks*sizeof(double));
    CUDA_ARRAY_ALLOC(reduce_buf_5, num_blocks*sizeof(double));
    CUDA_ARRAY_ALLOC(reduce_buf_6, num_blocks*sizeof(double));

    reduce_ptr_1 = thrust::device_ptr< double >(reduce_buf_1);
    reduce_ptr_2 = thrust::device_ptr< double >(reduce_buf_2);
    reduce_ptr_3 = thrust::device_ptr< double >(reduce_buf_3);
    reduce_ptr_4 = thrust::device_ptr< double >(reduce_buf_4);
    reduce_ptr_5 = thrust::device_ptr< double >(reduce_buf_5);
    reduce_ptr_6 = thrust::device_ptr< double >(reduce_buf_6);

    CUDA_ARRAY_ALLOC(pdv_reduce_array, num_blocks*sizeof(int));
    reduce_pdv = thrust::device_ptr< int >(pdv_reduce_array);

    thr_cellx = thrust::device_ptr< double >(cellx);
    thr_celly = thrust::device_ptr< double >(celly);
    thr_xvel0 = thrust::device_ptr< double >(xvel0);
    thr_yvel0 = thrust::device_ptr< double >(yvel0);
    thr_xvel1 = thrust::device_ptr< double >(xvel1);
    thr_yvel1 = thrust::device_ptr< double >(yvel1);
    thr_density0 = thrust::device_ptr< double >(density0);
    thr_energy0 = thrust::device_ptr< double >(energy0);
    thr_pressure = thrust::device_ptr< double >(pressure);
    thr_soundspeed = thrust::device_ptr< double >(soundspeed);

    CUDA_ARRAY_ALLOC(dev_left_send_buffer, sizeof(double)*(y_max+5)*2);
    CUDA_ARRAY_ALLOC(dev_right_send_buffer, sizeof(double)*(y_max+5)*2);
    CUDA_ARRAY_ALLOC(dev_top_send_buffer, sizeof(double)*(x_max+5)*2);
    CUDA_ARRAY_ALLOC(dev_bottom_send_buffer, sizeof(double)*(x_max+5)*2);

    CUDA_ARRAY_ALLOC(dev_left_recv_buffer, sizeof(double)*(y_max+5)*2);
    CUDA_ARRAY_ALLOC(dev_right_recv_buffer, sizeof(double)*(y_max+5)*2);
    CUDA_ARRAY_ALLOC(dev_top_recv_buffer, sizeof(double)*(x_max+5)*2);
    CUDA_ARRAY_ALLOC(dev_bottom_recv_buffer, sizeof(double)*(x_max+5)*2);

    #undef CUDA_ARRAY_ALLOC

#define ADD_BUFFER_DBG_MAP(name) arr_names[#name] = name;
    ADD_BUFFER_DBG_MAP(volume);
    ADD_BUFFER_DBG_MAP(soundspeed);
    ADD_BUFFER_DBG_MAP(pressure);
    ADD_BUFFER_DBG_MAP(viscosity);

    ADD_BUFFER_DBG_MAP(u);
    arr_names["p"] = work_array_1;

    ADD_BUFFER_DBG_MAP(density0);
    ADD_BUFFER_DBG_MAP(density1);
    ADD_BUFFER_DBG_MAP(energy0);
    ADD_BUFFER_DBG_MAP(energy1);
    ADD_BUFFER_DBG_MAP(xvel0);
    ADD_BUFFER_DBG_MAP(xvel1);
    ADD_BUFFER_DBG_MAP(yvel0);
    ADD_BUFFER_DBG_MAP(yvel1);
    ADD_BUFFER_DBG_MAP(xarea);
    ADD_BUFFER_DBG_MAP(yarea);
    ADD_BUFFER_DBG_MAP(vol_flux_x);
    ADD_BUFFER_DBG_MAP(vol_flux_y);
    ADD_BUFFER_DBG_MAP(mass_flux_x);
    ADD_BUFFER_DBG_MAP(mass_flux_y);

    ADD_BUFFER_DBG_MAP(cellx);
    ADD_BUFFER_DBG_MAP(celly);
    ADD_BUFFER_DBG_MAP(celldx);
    ADD_BUFFER_DBG_MAP(celldy);
    ADD_BUFFER_DBG_MAP(vertexx);
    ADD_BUFFER_DBG_MAP(vertexy);
    ADD_BUFFER_DBG_MAP(vertexdx);
    ADD_BUFFER_DBG_MAP(vertexdy);
#undef ADD_BUFFER_DBG_MAP
}

