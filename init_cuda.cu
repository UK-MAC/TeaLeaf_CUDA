#include "hip/hip_runtime.h"
/*Crown Copyright 2012 AWE.
 *
 * This file is part of TeaLeaf.
 *
 * TeaLeaf is free software: you can redistribute it and/or modify it under
 * the terms of the GNU General Public License as published by the
 * Free Software Foundation, either version 3 of the License, or (at your option)
 * any later version.
 *
 * TeaLeaf is distributed in the hope that it will be useful, but
 * WITHOUT ANY WARRANTY; without even the implied warranty of MERCHANTABILITY or
 * FITNESS FOR A PARTICULAR PURPOSE. See the GNU General Public License for more
 * details.
 *
 * You should have received a copy of the GNU General Public License along with
 * TeaLeaf. If not, see http://www.gnu.org/licenses/.
 */

/*
 *  @brief CUDA initialisation
 *  @author Michael Boulton NVIDIA Corporation
 *  @details Initialises CUDA devices and global storage
 */

#include "cuda_common.hpp"
#include "cuda_strings.hpp"

#include "mpi.h"
#include <sstream>
#include <cstdio>
#include <cassert>

TealeafCudaChunk cuda_chunk;

extern "C" void initialise_cuda_
(INITIALISE_ARGS)
{
    cuda_chunk = TealeafCudaChunk(in_x_min,
                                in_x_max,
                                in_y_min,
                                in_y_max);
}

TealeafCudaChunk::TealeafCudaChunk
(void)
{
    ;
}

TealeafCudaChunk::TealeafCudaChunk
(INITIALISE_ARGS)
:x_min(*in_x_min),
x_max(*in_x_max),
y_min(*in_y_min),
y_max(*in_y_max)
{
    // FIXME (and opencl really)
    // make a better platform agnostic way of selecting devices

    MPI_Comm_rank(MPI_COMM_WORLD, &rank);

    // Read in from file - easier than passing in from fortran
    std::ifstream input("tea.in");
    input.exceptions(std::ifstream::badbit);

    if (!input.is_open())
    {
        // should never happen
        DIE("Input file not found\n");
    }

    profiler_on = paramEnabled(input, "profiler_on");

    int device_id = readInt(input, "cuda_device");
    device_id = (device_id < 0) ? 0 : device_id;

    hipDeviceReset();

    // account for MPI
    int num_devices;
    hipGetDeviceCount(&num_devices);

    if (num_devices < device_id)
    {
        DIE("Device id %d specified in tea.in, but only %d devices available", device_id, num_devices);
    }

    int err = hipSetDevice(device_id);

    if (err != hipSuccess)
    {
        DIE("Setting device id to %d in rank %d failed with error code %d\n", device_id, rank, err);
    }

    struct hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, device_id);
    std::cout << "CUDA in rank " << rank << " using " << prop.name << std::endl;

    int file_halo_depth = readInt(input, "halo_depth");
    halo_exchange_depth = file_halo_depth;

    if (halo_exchange_depth < 1)
    {
        DIE("Halo exchange depth unspecified or was too small");
    }

    bool tl_use_jacobi = paramEnabled(input, "tl_use_jacobi");
    bool tl_use_cg = paramEnabled(input, "tl_use_cg");
    bool tl_use_chebyshev = paramEnabled(input, "tl_use_chebyshev");
    bool tl_use_ppcg = paramEnabled(input, "tl_use_ppcg");

    // set solve
    if(!rank)fprintf(stdout, "Solver to use: ");
    if (tl_use_ppcg)
    {
        tea_solver = TEA_ENUM_PPCG;
        if(!rank)fprintf(stdout, "PPCG\n");
    }
    else if (tl_use_chebyshev)
    {
        tea_solver = TEA_ENUM_CHEBYSHEV;
        if(!rank)fprintf(stdout, "Chebyshev + CG\n");
    }
    else if (tl_use_cg)
    {
        tea_solver = TEA_ENUM_CG;
        if(!rank)fprintf(stdout, "Conjugate gradient\n");
    }
    else if (tl_use_jacobi)
    {
        tea_solver = TEA_ENUM_JACOBI;
        if(!rank)fprintf(stdout, "Jacobi\n");
    }
    else
    {
        tea_solver = TEA_ENUM_JACOBI;
        if(!rank)fprintf(stdout, "Jacobi (no solver specified in tea.in)\n");
    }

    std::string desired_preconditioner = readString(input, "tl_preconditioner_type");

    // set preconditioner type
    if(!rank)fprintf(stdout, "Preconditioner to use: ");
    if (desired_preconditioner.find("jac_diag") != std::string::npos)
    {
        preconditioner_type = TL_PREC_JAC_DIAG;
        if(!rank)fprintf(stdout, "Diagonal Jacobi\n");
    }
    else if (desired_preconditioner.find("jac_block") != std::string::npos)
    {
        preconditioner_type = TL_PREC_JAC_BLOCK;
        if(!rank)fprintf(stdout, "Block Jacobi\n");
    }
    else if (desired_preconditioner.find("none") != std::string::npos)
    {
        preconditioner_type = TL_PREC_NONE;
        if(!rank)fprintf(stdout, "None\n");
    }
    else
    {
        preconditioner_type = TL_PREC_NONE;
        if(!rank)fprintf(stdout, "None (no preconditioner specified in tea.in)\n");
    }

    initSizes();
    initBuffers();
}

void TealeafCudaChunk::initSizes
(void)
{
    grid_dim = dim3(
        std::ceil((x_max + 2.0*halo_exchange_depth)/LOCAL_X),
        std::ceil((y_max + 2.0*halo_exchange_depth)/LOCAL_Y));
    num_blocks = grid_dim.x*grid_dim.y;

    #define UPDATE_HALO_SIZE 32

    for (int depth = 1; depth <= 2; depth++)
    {
        update_bt_block_sizes[depth] = dim3(UPDATE_HALO_SIZE, 1);
        update_lr_block_sizes[depth] = dim3(1, UPDATE_HALO_SIZE);
    }

    update_bt_block_sizes[halo_exchange_depth] = update_bt_block_sizes[1];
    update_lr_block_sizes[halo_exchange_depth] = update_lr_block_sizes[1];

    std::map<int, dim3>::iterator typedef irangeit;
    for (irangeit key = update_lr_block_sizes.begin();
        key != update_lr_block_sizes.end(); key++)
    {
        int depth = key->first;

        int min_update_bt_grid_dim = x_max + 2*depth;
        int min_update_lr_grid_dim = y_max + 2*depth;

        int num_blocks_bt = 1;
        int num_blocks_lr = 1;

        while (update_bt_block_sizes[depth].x*num_blocks_bt < min_update_bt_grid_dim)
            num_blocks_bt++;
        while (update_lr_block_sizes[depth].y*num_blocks_lr < min_update_lr_grid_dim)
            num_blocks_lr++;

        update_bt_num_blocks[depth] = dim3(num_blocks_bt, depth);
        update_lr_num_blocks[depth] = dim3(depth, num_blocks_lr);
    }

    kernel_info_t kernel_info_generic;

    kernel_info_generic.x_min = x_min;
    kernel_info_generic.x_max = x_max;
    kernel_info_generic.y_min = y_min;
    kernel_info_generic.y_max = y_max;

    kernel_info_generic.halo_depth = halo_exchange_depth;
    kernel_info_generic.preconditioner_type = preconditioner_type;

    kernel_info_generic.x_offset = halo_exchange_depth;
    kernel_info_generic.y_offset = halo_exchange_depth;

    kernel_info_map["device_initialise_chunk_kernel"] = kernel_info_t(kernel_info_generic, -halo_exchange_depth, halo_exchange_depth, -halo_exchange_depth, halo_exchange_depth);

    kernel_info_map["device_initialise_chunk_kernel_vertex"] = kernel_info_t(kernel_info_generic, -halo_exchange_depth, halo_exchange_depth, -halo_exchange_depth, halo_exchange_depth);
    kernel_info_map["device_generate_chunk_init"] = kernel_info_t(kernel_info_generic, 0, 0, 0, 0);
    kernel_info_map["device_generate_chunk_kernel"] = kernel_info_t(kernel_info_generic, 0, 0, 0, 0);
    kernel_info_map["device_generate_chunk_init_u"] = kernel_info_t(kernel_info_generic, 0, 0, 0, 0);
    kernel_info_map["device_generate_chunk"] = kernel_info_t(kernel_info_generic, 0, 0, 0, 0);

    kernel_info_map["device_set_field_kernel"] = kernel_info_t(kernel_info_generic, 0, 0, 0, 0);
    kernel_info_map["device_field_summary_kernel"] = kernel_info_t(kernel_info_generic, 0, 0, 0, 0);

    kernel_info_map["device_update_halo_top"] = kernel_info_t(kernel_info_generic, 0, 0, 0, 0);
    kernel_info_map["device_update_halo_bottom"] = kernel_info_t(kernel_info_generic, 0, 0, 0, 0);
    kernel_info_map["device_update_halo_left"] = kernel_info_t(kernel_info_generic, 0, 0, 0, 0);
    kernel_info_map["device_update_halo_right"] = kernel_info_t(kernel_info_generic, 0, 0, 0, 0);

    kernel_info_map["device_pack_left_buffer"] = kernel_info_t(kernel_info_generic, 0, 0, 0, 0);
    kernel_info_map["device_unpack_left_buffer"] = kernel_info_t(kernel_info_generic, 0, 0, 0, 0);
    kernel_info_map["device_pack_right_buffer"] = kernel_info_t(kernel_info_generic, 0, 0, 0, 0);
    kernel_info_map["device_unpack_right_buffer"] = kernel_info_t(kernel_info_generic, 0, 0, 0, 0);
    kernel_info_map["device_pack_bottom_buffer"] = kernel_info_t(kernel_info_generic, 0, 0, 0, 0);
    kernel_info_map["device_unpack_bottom_buffer"] = kernel_info_t(kernel_info_generic, 0, 0, 0, 0);
    kernel_info_map["device_pack_top_buffer"] = kernel_info_t(kernel_info_generic, 0, 0, 0, 0);
    kernel_info_map["device_unpack_top_buffer"] = kernel_info_t(kernel_info_generic, 0, 0, 0, 0);

    if (tea_solver == TEA_ENUM_CG ||
    tea_solver == TEA_ENUM_CHEBYSHEV ||
    tea_solver == TEA_ENUM_PPCG)
    {
        kernel_info_map["device_tea_leaf_cg_solve_calc_w"] = kernel_info_t(kernel_info_generic, 0, 0, 0, 0);
        kernel_info_map["device_tea_leaf_cg_solve_calc_ur"] = kernel_info_t(kernel_info_generic, 0, 0, 0, 0);
        kernel_info_map["device_tea_leaf_cg_solve_calc_p"] = kernel_info_t(kernel_info_generic, 0, 0, 0, 0);
        kernel_info_map["device_tea_leaf_cg_solve_init_p"] = kernel_info_t(kernel_info_generic, 0, 0, 0, 0);

        if (tea_solver == TEA_ENUM_CHEBYSHEV)
        {
            kernel_info_map["device_tea_leaf_cheby_solve_init_p"] = kernel_info_t(kernel_info_generic, 0, 0, 0, 0);
            kernel_info_map["device_tea_leaf_cheby_solve_calc_u"] = kernel_info_t(kernel_info_generic, 0, 0, 0, 0);
            kernel_info_map["device_tea_leaf_cheby_solve_calc_p"] = kernel_info_t(kernel_info_generic, 0, 0, 0, 0);
        }
        else if (tea_solver == TEA_ENUM_PPCG)
        {
            kernel_info_map["device_tea_leaf_ppcg_solve_init_sd"] = kernel_info_t(kernel_info_generic, 0, 0, 0, 0);
            kernel_info_map["device_tea_leaf_ppcg_solve_calc_sd"] = kernel_info_t(kernel_info_generic, 0, 0, 0, 0);
            kernel_info_map["device_tea_leaf_ppcg_solve_update_r"] = kernel_info_t(kernel_info_generic, 0, 0, 0, 0);
            kernel_info_map["device_tea_leaf_ppcg_solve_calc_sd_new"] = kernel_info_t(kernel_info_generic, 0, 0, 0, 0);
            kernel_info_map["device_tea_leaf_ppcg_solve_update_r_new"] = kernel_info_t(kernel_info_generic, 0, 0, 0, 0);            
            kernel_info_map["device_tea_leaf_ppcg_store_r"] = kernel_info_t(kernel_info_generic, 0, 0, 0, 0);
            kernel_info_map["device_tea_leaf_ppcg_update_z"] = kernel_info_t(kernel_info_generic, 0, 0, 0, 0);            
            kernel_info_map["device_tea_leaf_calc_rrn"] = kernel_info_t(kernel_info_generic, 0, 0, 0, 0);
            kernel_info_map["device_tea_leaf_ppcg_solve_init_sd_new"] = kernel_info_t(kernel_info_generic, 0, 0, 0, 0); 
            kernel_info_map["device_tea_leaf_ppcg_solve_init"] = kernel_info_t(kernel_info_generic, 0, 0, 0, 0);
        kernel_info_map["device_tea_leaf_ppcg_solve_calc_p"] = kernel_info_t(kernel_info_generic, 0, 0, 0, 0);                                                  
        }
    }
    else
    {
        kernel_info_map["device_tea_leaf_jacobi_copy_u"] = kernel_info_t(kernel_info_generic, 0, 0, 0, 0);
        kernel_info_map["device_tea_leaf_jacobi_solve"] = kernel_info_t(kernel_info_generic, 0, 0, 0, 0);
    }

    kernel_info_map["device_tea_leaf_finalise"] = kernel_info_t(kernel_info_generic, 0, 0, 0, 0);
    kernel_info_map["device_tea_leaf_calc_residual"] = kernel_info_t(kernel_info_generic, 0, 0, 0, 0);
    kernel_info_map["device_tea_leaf_calc_2norm"] = kernel_info_t(kernel_info_generic, 0, 0, 0, 0);

    kernel_info_map["device_tea_leaf_block_init"] = kernel_info_t(kernel_info_generic, 0, 0, 0, 0);
    kernel_info_map["device_tea_leaf_block_solve"] = kernel_info_t(kernel_info_generic, 0, 0, 0, 0);

    kernel_info_map["device_tea_leaf_init_common"] = kernel_info_t(kernel_info_generic, 0, halo_exchange_depth,0, halo_exchange_depth);
    kernel_info_map["device_tea_leaf_zero_boundaries"] = kernel_info_t(kernel_info_generic, -halo_exchange_depth, halo_exchange_depth, -halo_exchange_depth, halo_exchange_depth);
    kernel_info_map["device_tea_leaf_init_jac_diag"] = kernel_info_t(kernel_info_generic, -halo_exchange_depth, halo_exchange_depth, -halo_exchange_depth, halo_exchange_depth);
}

void TealeafCudaChunk::initBuffers
(void)
{
    #define CUDA_ARRAY_ALLOC(arr, size)     \
            hipMalloc((void**) &arr, size);\
            errorHandler(__LINE__, __FILE__);\
            hipDeviceSynchronize();        \
            hipMemset(arr, 0, size);       \
            hipDeviceSynchronize();        \
            CUDA_ERR_CHECK;

    // number of bytes to allocate for x size array
    #define BUFSZX(x_extra)   \
        ( ((x_max) + 2*halo_exchange_depth + x_extra)       \
        * sizeof(double) )

    // number of bytes to allocate for y size array
    #define BUFSZY(y_extra)   \
        ( ((y_max) + 2*halo_exchange_depth + y_extra)       \
        * sizeof(double) )

    // number of bytes to allocate for 2d array
    #define BUFSZ2D(x_extra, y_extra)   \
        ( ((x_max) + 2*halo_exchange_depth + x_extra)       \
        * ((y_max) + 2*halo_exchange_depth + y_extra)       \
        * sizeof(double) )

    CUDA_ARRAY_ALLOC(volume, BUFSZ2D(0, 0));
    CUDA_ARRAY_ALLOC(soundspeed, BUFSZ2D(0, 0));

    CUDA_ARRAY_ALLOC(tri_cp, BUFSZ2D(0, 0));
    CUDA_ARRAY_ALLOC(tri_bfp, BUFSZ2D(0, 0));

    CUDA_ARRAY_ALLOC(density, BUFSZ2D(0, 0));
    CUDA_ARRAY_ALLOC(energy0, BUFSZ2D(0, 0));
    CUDA_ARRAY_ALLOC(energy1, BUFSZ2D(0, 0));

    CUDA_ARRAY_ALLOC(xarea, BUFSZ2D(1, 0));
    CUDA_ARRAY_ALLOC(yarea, BUFSZ2D(0, 1));

    CUDA_ARRAY_ALLOC(cellx, BUFSZX(0)); 
    CUDA_ARRAY_ALLOC(celldx, BUFSZX(0));
    CUDA_ARRAY_ALLOC(vertexx, BUFSZX(1));
    CUDA_ARRAY_ALLOC(vertexdx, BUFSZX(1));

    CUDA_ARRAY_ALLOC(celly, BUFSZY(0));
    CUDA_ARRAY_ALLOC(celldy, BUFSZY(0));
    CUDA_ARRAY_ALLOC(vertexy, BUFSZY(1));
    CUDA_ARRAY_ALLOC(vertexdy, BUFSZY(1));

    CUDA_ARRAY_ALLOC(u, BUFSZ2D(0, 0));
    CUDA_ARRAY_ALLOC(u0, BUFSZ2D(0, 0));
    CUDA_ARRAY_ALLOC(vector_z, BUFSZ2D(0, 0));

    CUDA_ARRAY_ALLOC(vector_p, BUFSZ2D(0, 0));
    CUDA_ARRAY_ALLOC(vector_r, BUFSZ2D(0, 0));
    CUDA_ARRAY_ALLOC(vector_w, BUFSZ2D(0, 0));
    CUDA_ARRAY_ALLOC(vector_Mi, BUFSZ2D(0, 0));
    CUDA_ARRAY_ALLOC(vector_Kx, BUFSZ2D(0, 0));
    CUDA_ARRAY_ALLOC(vector_Ky, BUFSZ2D(0, 0));
    CUDA_ARRAY_ALLOC(vector_sd, BUFSZ2D(0, 0));

// For PPCG

    CUDA_ARRAY_ALLOC(vector_rtemp, BUFSZ2D(0, 0));
    CUDA_ARRAY_ALLOC(vector_utemp, BUFSZ2D(0, 0));
    CUDA_ARRAY_ALLOC(vector_r_store, BUFSZ2D(0, 0));

    CUDA_ARRAY_ALLOC(left_buffer, (y_max+2*halo_exchange_depth)*halo_exchange_depth*NUM_BUFFERED_FIELDS*sizeof(double));
    CUDA_ARRAY_ALLOC(right_buffer, (y_max+2*halo_exchange_depth)*halo_exchange_depth*NUM_BUFFERED_FIELDS*sizeof(double));
    CUDA_ARRAY_ALLOC(bottom_buffer, (x_max+2*halo_exchange_depth)*halo_exchange_depth*NUM_BUFFERED_FIELDS*sizeof(double));
    CUDA_ARRAY_ALLOC(top_buffer, (x_max+2*halo_exchange_depth)*halo_exchange_depth*NUM_BUFFERED_FIELDS*sizeof(double));

    CUDA_ARRAY_ALLOC(reduce_buf_1, num_blocks*sizeof(double));
    CUDA_ARRAY_ALLOC(reduce_buf_2, num_blocks*sizeof(double));
    CUDA_ARRAY_ALLOC(reduce_buf_3, num_blocks*sizeof(double));
    CUDA_ARRAY_ALLOC(reduce_buf_4, num_blocks*sizeof(double));

    // To make sure memory is allocated later on
    ch_alphas_device = NULL;
    ch_betas_device = NULL;

    #undef CUDA_ARRAY_ALLOC

#define ADD_BUFFER_DBG_MAP(name) arr_names[#name] = name;
    ADD_BUFFER_DBG_MAP(volume);
    ADD_BUFFER_DBG_MAP(soundspeed);

    ADD_BUFFER_DBG_MAP(u);
    ADD_BUFFER_DBG_MAP(u0);

    ADD_BUFFER_DBG_MAP(vector_p);
    ADD_BUFFER_DBG_MAP(vector_r);
    ADD_BUFFER_DBG_MAP(vector_w);
    ADD_BUFFER_DBG_MAP(vector_sd);
    ADD_BUFFER_DBG_MAP(vector_Mi);
    ADD_BUFFER_DBG_MAP(vector_Kx);
    ADD_BUFFER_DBG_MAP(vector_Ky);
    // PPCG
    ADD_BUFFER_DBG_MAP(vector_rtemp);
    ADD_BUFFER_DBG_MAP(vector_utemp);
    ADD_BUFFER_DBG_MAP(vector_r_store);    

    ADD_BUFFER_DBG_MAP(density);
    ADD_BUFFER_DBG_MAP(energy0);
    ADD_BUFFER_DBG_MAP(energy1);
    ADD_BUFFER_DBG_MAP(xarea);
    ADD_BUFFER_DBG_MAP(yarea);

    ADD_BUFFER_DBG_MAP(cellx);
    ADD_BUFFER_DBG_MAP(celly);
    ADD_BUFFER_DBG_MAP(celldx);
    ADD_BUFFER_DBG_MAP(celldy);
    ADD_BUFFER_DBG_MAP(vertexx);
    ADD_BUFFER_DBG_MAP(vertexy);
    ADD_BUFFER_DBG_MAP(vertexdx);
    ADD_BUFFER_DBG_MAP(vertexdy);
#undef ADD_BUFFER_DBG_MAP
}

