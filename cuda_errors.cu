#if defined(MPI_HDR)
extern "C" void clover_abort_();
#endif

#include "cuda_common.hpp"
#include <cstdarg>
#include <cstdio>

static const char* errorCodes
(int err_code)
{
    switch(err_code)
    {
        case hipSuccess: return "hipSuccess"; // 0
        case hipErrorMissingConfiguration: return "hipErrorMissingConfiguration"; // 1
        case hipErrorOutOfMemory: return "hipErrorOutOfMemory"; // 2
        case hipErrorNotInitialized: return "hipErrorNotInitialized"; // 3
        case hipErrorLaunchFailure: return "hipErrorLaunchFailure"; // 4
        case hipErrorPriorLaunchFailure: return "hipErrorPriorLaunchFailure"; // 5
        case hipErrorLaunchTimeOut: return "hipErrorLaunchTimeOut"; // 6
        case hipErrorLaunchOutOfResources: return "hipErrorLaunchOutOfResources"; // 7
        case hipErrorInvalidDeviceFunction: return "hipErrorInvalidDeviceFunction"; // 8
        case hipErrorInvalidConfiguration: return "hipErrorInvalidConfiguration"; // 9
        case hipErrorInvalidDevice: return "hipErrorInvalidDevice"; // 10
        case hipErrorInvalidValue: return "hipErrorInvalidValue";// 11
        case hipErrorInvalidPitchValue: return "hipErrorInvalidPitchValue";// 12
        case hipErrorInvalidSymbol: return "hipErrorInvalidSymbol";// 13
        case hipErrorMapFailed: return "hipErrorMapFailed";// 14
        case hipErrorUnmapFailed: return "hipErrorUnmapFailed";// 15
        case cudaErrorInvalidHostPointer: return "cudaErrorInvalidHostPointer";// 16
        case hipErrorInvalidDevicePointer: return "hipErrorInvalidDevicePointer";// 17
        case hipErrorInvalidTexture: return "hipErrorInvalidTexture";// 18
        case cudaErrorInvalidTextureBinding: return "cudaErrorInvalidTextureBinding";// 19
        case hipErrorInvalidChannelDescriptor: return "hipErrorInvalidChannelDescriptor";// 20
        case hipErrorInvalidMemcpyDirection: return "hipErrorInvalidMemcpyDirection";// 21
        case cudaErrorAddressOfConstant: return "cudaErrorAddressOfConstant";// 22
        case cudaErrorTextureFetchFailed: return "cudaErrorTextureFetchFailed";// 23
        case cudaErrorTextureNotBound: return "cudaErrorTextureNotBound";// 24
        case cudaErrorSynchronizationError: return "cudaErrorSynchronizationError";// 25
        case cudaErrorInvalidFilterSetting: return "cudaErrorInvalidFilterSetting";// 26
        case cudaErrorInvalidNormSetting: return "cudaErrorInvalidNormSetting";// 27
        case cudaErrorMixedDeviceExecution: return "cudaErrorMixedDeviceExecution";// 28
        case hipErrorDeinitialized: return "hipErrorDeinitialized";// 29
        case hipErrorUnknown: return "hipErrorUnknown";// 30
        case cudaErrorNotYetImplemented: return "cudaErrorNotYetImplemented";// 31
        case cudaErrorMemoryValueTooLarge: return "cudaErrorMemoryValueTooLarge";// 32
        case hipErrorInvalidHandle: return "hipErrorInvalidHandle";// 33
        case hipErrorNotReady: return "hipErrorNotReady";// 34
        case hipErrorInsufficientDriver: return "hipErrorInsufficientDriver";// 35
        case hipErrorSetOnActiveProcess: return "hipErrorSetOnActiveProcess";// 36
        case cudaErrorInvalidSurface: return "cudaErrorInvalidSurface";// 37
        case hipErrorNoDevice: return "hipErrorNoDevice";// 38
        case hipErrorECCNotCorrectable: return "hipErrorECCNotCorrectable";// 39
        case hipErrorSharedObjectSymbolNotFound: return "hipErrorSharedObjectSymbolNotFound";// 40
        case hipErrorSharedObjectInitFailed: return "hipErrorSharedObjectInitFailed";// 41
        case hipErrorUnsupportedLimit: return "hipErrorUnsupportedLimit";// 42
        case cudaErrorDuplicateVariableName: return "cudaErrorDuplicateVariableName";// 43
        case cudaErrorDuplicateTextureName: return "cudaErrorDuplicateTextureName";// 44
        case cudaErrorDuplicateSurfaceName: return "cudaErrorDuplicateSurfaceName";// 45
        case cudaErrorDevicesUnavailable: return "cudaErrorDevicesUnavailable";// 46
        case hipErrorInvalidImage: return "hipErrorInvalidImage";// 47
        case hipErrorNoBinaryForGpu: return "hipErrorNoBinaryForGpu";// 48
        case cudaErrorIncompatibleDriverContext: return "cudaErrorIncompatibleDriverContext";// 49
        case hipErrorPeerAccessAlreadyEnabled: return "hipErrorPeerAccessAlreadyEnabled";// 50
        case hipErrorPeerAccessNotEnabled: return "hipErrorPeerAccessNotEnabled";// 51
        case hipErrorContextAlreadyInUse: return "hipErrorContextAlreadyInUse";// 52
        case hipErrorProfilerDisabled: return "hipErrorProfilerDisabled";// 53
        case hipErrorProfilerNotInitialized: return "hipErrorProfilerNotInitialized";// 54
        case hipErrorProfilerAlreadyStarted: return "hipErrorProfilerAlreadyStarted";// 55
        case hipErrorProfilerAlreadyStopped: return "hipErrorProfilerAlreadyStopped";// 56
        case hipErrorAssert: return "hipErrorAssert";// 57
        case cudaErrorTooManyPeers: return "cudaErrorTooManyPeers";// 58
        case hipErrorHostMemoryAlreadyRegistered: return "hipErrorHostMemoryAlreadyRegistered";// 59
        case hipErrorHostMemoryNotRegistered: return "hipErrorHostMemoryNotRegistered";// 60
        case hipErrorOperatingSystem: return "hipErrorOperatingSystem";// 61
        case cudaErrorStartupFailure: return "cudaErrorStartupFailure";// 62
        case cudaErrorApiFailureBase: return "cudaErrorApiFailureBase";// 63
        default: return "Unknown error";
    }
}

void CloverleafCudaChunk::errorHandler
(int line_num, const char* file)
{
    hipDeviceSynchronize();
    int l_e = hipGetLastError();
    if (hipSuccess != l_e)
    {
        cloverDie(line_num, file, "Error in %s - return code %d (%s)\n", file, l_e, errorCodes(l_e));
    }
}

// print out timing info when done
CloverleafCudaChunk::~CloverleafCudaChunk
(void)
{
    if (profiler_on)
    {
        fprintf(stdout, "@@@@@ PROFILING @@@@@\n");

        for (std::map<std::string, double>::iterator ii = kernel_times.begin();
            ii != kernel_times.end(); ii++)
        {
            fprintf(stdout, "%35s : %.3f\n", ii->first.c_str(), ii->second);
        }
    }
}

std::vector<double> CloverleafCudaChunk::dumpArray
(const std::string& arr_name, int x_extra, int y_extra)
{
    std::vector<double> host_arr(BUFSZ2D(x_extra, y_extra)/sizeof(double));

    hipDeviceSynchronize();

    try
    {
        hipMemcpy(&host_arr.front(), arr_names.at(arr_name),
            BUFSZ2D(x_extra, y_extra), hipMemcpyDeviceToHost);
    }
    catch (std::out_of_range e)
    {
        DIE("Error - %s was not in the arr_names map\n", arr_name.c_str());
    }

    errorHandler(__LINE__, __FILE__);

    return host_arr;
}

// called when something goes wrong
void CloverleafCudaChunk::cloverDie
(int line, const char* filename, const char* format, ...)
{
    fprintf(stderr, "@@@@@\n");
    fprintf(stderr, "\x1b[31m");
    fprintf(stderr, "Fatal error at line %d in %s:", line, filename);
    fprintf(stderr, "\x1b[0m");
    fprintf(stderr, "\n");

    va_list arglist;
    va_start(arglist, format);
    vfprintf(stderr, format, arglist);
    va_end(arglist);

    // TODO add logging or something

    fprintf(stderr, "\nExiting\n");

#if defined(MPI_HDR)
    clover_abort_();
#else
    exit(1);
#endif
}

