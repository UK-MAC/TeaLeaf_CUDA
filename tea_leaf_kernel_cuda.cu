#include "hip/hip_runtime.h"
#include "cuda_common.hpp"

#define CG_DO_PRECONDITION

#include "kernel_files/tea_leaf_common.cuknl"
#include "kernel_files/tea_leaf_jacobi.cuknl"
#include "kernel_files/tea_leaf_cg.cuknl"
#include "kernel_files/tea_leaf_cheby.cuknl"

#include <cassert>

// same as in fortran
#define CONDUCTIVITY 1
#define RECIP_CONDUCTIVITY 2

// copy back dx/dy and calculate rx/ry
void CloverleafCudaChunk::calcrxry
(double dt, double * rx, double * ry)
{
    static int initd = 0;
    if (!initd)
    {
        // FIXME remove this check - only relaly done once, one sync doesnt do much anyway
        // make sure intialise chunk has finished
        hipDeviceSynchronize();
        // celldx doesnt change after that so check once
        initd = 1;
    }

    double dx, dy;

    hipMemcpy(&dx, celldx, sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(&dy, celldy, sizeof(double), hipMemcpyDeviceToHost);

    CUDA_ERR_CHECK;

    *rx = dt/(dx*dx);
    *ry = dt/(dy*dy);
}

/********************/

// Chebyshev solver
extern "C" void tea_leaf_kernel_cheby_copy_u_cuda_
(double* rro)
{
    cuda_chunk.tea_leaf_cheby_copy_u(rro);
}

extern "C" void tea_leaf_calc_2norm_kernel_cuda_
(int* norm_array, double* norm)
{
    cuda_chunk.tea_leaf_calc_2norm_kernel(*norm_array, norm);
}

extern "C" void tea_leaf_kernel_cheby_init_cuda_
(const double * ch_alphas, const double * ch_betas, int* n_coefs,
 const double * rx, const double * ry, const double * theta, double* error)
{
    cuda_chunk.tea_leaf_kernel_cheby_init(ch_alphas, ch_betas, *n_coefs,
        *rx, *ry, *theta, error);
}

extern "C" void tea_leaf_kernel_cheby_iterate_cuda_
(const double * ch_alphas, const double * ch_betas, int *n_coefs,
 const double * rx, const double * ry, const int * cheby_calc_step)
{
    cuda_chunk.tea_leaf_kernel_cheby_iterate(ch_alphas, ch_betas, *n_coefs,
        *rx, *ry, *cheby_calc_step);
}

void CloverleafCudaChunk::tea_leaf_cheby_copy_u
(double* rro)
{
    hipDeviceSynchronize();
    hipMemcpy(u0, u, BUFSZ2D(0, 0), hipMemcpyDeviceToDevice);
    *rro = thrust::reduce(reduce_ptr_2, reduce_ptr_2 + num_blocks, 0.0);
}

void CloverleafCudaChunk::tea_leaf_calc_2norm_kernel
(int norm_array, double* norm)
{
    if (norm_array == 0)
    {
        // norm of u0
        device_tea_leaf_cheby_solve_calc_resid<<< num_blocks, BLOCK_SZ >>>
        (x_min, x_max, y_min, y_max, u0, reduce_buf_1);
    }
    else if (norm_array == 1)
    {
        // norm of r
        device_tea_leaf_cheby_solve_calc_resid<<< num_blocks, BLOCK_SZ >>>
        (x_min, x_max, y_min, y_max, work_array_2, reduce_buf_1);
    }
    else
    {
        DIE("Invalid value '%d' for norm_array passed, should be [1, 2]", norm_array);
    }

    CUDA_ERR_CHECK;

    *norm = thrust::reduce(reduce_ptr_1, reduce_ptr_1 + num_blocks, 0.0);
}

void CloverleafCudaChunk::tea_leaf_kernel_cheby_init
(const double * ch_alphas, const double * ch_betas, int n_coefs,
 const double rx, const double ry, const double theta, double* error)
{
    assert(tea_solver == TEA_ENUM_CHEBYSHEV);

    size_t ch_buf_sz = n_coefs*sizeof(double);

    // upload to device
    hipMalloc((void**) &ch_alphas_device, ch_buf_sz);
    hipMalloc((void**) &ch_betas_device, ch_buf_sz);
    hipMemcpy(ch_alphas_device, ch_alphas, ch_buf_sz, hipMemcpyHostToDevice);
    hipMemcpy(ch_betas_device, ch_betas, ch_buf_sz, hipMemcpyHostToDevice);

    CUDA_ERR_CHECK;

    // this will junk p but we don't need it anyway
    tea_leaf_kernel_cheby_iterate(ch_alphas, ch_betas, 0, rx, ry, 1);

    // then correct p
    CUDALAUNCH(device_tea_leaf_cheby_solve_init_p, work_array_1, z, theta);

    // do a step like in fortran
    tea_leaf_kernel_cheby_iterate(ch_alphas, ch_betas, 0, rx, ry, 1);

    // get norm of r
    tea_leaf_calc_2norm_kernel(1, error);
}

void CloverleafCudaChunk::tea_leaf_kernel_cheby_iterate
(const double * ch_alphas, const double * ch_betas, int n_coefs,
 const double rx, const double ry, const int cheby_calc_step)
{
    CUDALAUNCH(device_tea_leaf_cheby_solve_calc_u, u, work_array_1);

    CUDALAUNCH(device_tea_leaf_cheby_solve_calc_p, u, u0,
        work_array_1, work_array_2, work_array_3, work_array_4,
        z, work_array_5, work_array_6,
        ch_alphas_device, ch_betas_device,
        rx, ry, cheby_calc_step-1);
}

/********************/

// CG solver functions
extern "C" void tea_leaf_kernel_init_cg_cuda_
(const int * coefficient, double * dt, double * rx, double * ry, double * rro)
{
    cuda_chunk.tea_leaf_init_cg(*coefficient, *dt, rx, ry, rro);
}

extern "C" void tea_leaf_kernel_solve_cg_cuda_calc_w_
(const double * rx, const double * ry, double * pw)
{
    cuda_chunk.tea_leaf_kernel_cg_calc_w(*rx, *ry, pw);
}
extern "C" void tea_leaf_kernel_solve_cg_cuda_calc_ur_
(double * alpha, double * rrn)
{
    cuda_chunk.tea_leaf_kernel_cg_calc_ur(*alpha, rrn);
}
extern "C" void tea_leaf_kernel_solve_cg_cuda_calc_p_
(double * beta)
{
    cuda_chunk.tea_leaf_kernel_cg_calc_p(*beta);
}

/********************/

void CloverleafCudaChunk::tea_leaf_init_cg
(int coefficient, double dt, double * rx, double * ry, double * rro)
{
    if (coefficient != CONDUCTIVITY && coefficient != RECIP_CONDUCTIVITY)
    {
        DIE("Unknown coefficient %d passed to tea leaf\n", coefficient);
    }

    assert(tea_solver == TEA_ENUM_CG || tea_solver == TEA_ENUM_CHEBYSHEV);

    calcrxry(dt, rx, ry);

    CUDALAUNCH(device_tea_leaf_cg_init_u, density1, energy1, u,
        work_array_1, work_array_2, work_array_3, coefficient);

    // init Kx, Ky
    CUDALAUNCH(device_tea_leaf_cg_init_directions, work_array_3, work_array_5, work_array_6);

    // premultiply Kx/Ky
    CUDALAUNCH(device_tea_leaf_init_diag, work_array_5, work_array_6, *rx, *ry);

    // get initial guess in w, r, etc
    CUDALAUNCH(device_tea_leaf_cg_init_others, reduce_buf_2, u,
        work_array_1, work_array_2, work_array_3, work_array_4, z,
        work_array_5, work_array_6, *rx, *ry);

    *rro = thrust::reduce(reduce_ptr_2, reduce_ptr_2 + num_blocks, 0.0);
}

void CloverleafCudaChunk::tea_leaf_kernel_cg_calc_w
(double rx, double ry, double* pw)
{
    CUDALAUNCH(device_tea_leaf_cg_solve_calc_w, reduce_buf_3,
        work_array_1, work_array_4, work_array_5, work_array_6, rx, ry);

    *pw = thrust::reduce(reduce_ptr_3, reduce_ptr_3 + num_blocks, 0.0);
}

void CloverleafCudaChunk::tea_leaf_kernel_cg_calc_ur
(double alpha, double* rrn)
{
    CUDALAUNCH(device_tea_leaf_cg_solve_calc_ur, alpha, reduce_buf_4, u, work_array_1,
        work_array_2, work_array_4, z, work_array_3);

    *rrn = thrust::reduce(reduce_ptr_4, reduce_ptr_4 + num_blocks, 0.0);
}

void CloverleafCudaChunk::tea_leaf_kernel_cg_calc_p
(double beta)
{
    CUDALAUNCH(device_tea_leaf_cg_solve_calc_p, beta, work_array_1, work_array_2, z);
}

/********************/

// jacobi solver functions
extern "C" void tea_leaf_kernel_init_cuda_
(const int * coefficient, double * dt, double * rx, double * ry)
{
    cuda_chunk.tea_leaf_init_jacobi(*coefficient, *dt, rx, ry);
}

extern "C" void tea_leaf_kernel_solve_cuda_
(const double * rx, const double * ry, double * error)
{
    cuda_chunk.tea_leaf_kernel_jacobi(*rx, *ry, error);
}

// jacobi
void CloverleafCudaChunk::tea_leaf_init_jacobi
(int coefficient, double dt, double * rx, double * ry)
{
    if (coefficient != CONDUCTIVITY && coefficient != RECIP_CONDUCTIVITY)
    {
        DIE("Unknown coefficient %d passed to tea leaf\n", coefficient);
    }

    calcrxry(dt, rx, ry);

    CUDALAUNCH(device_tea_leaf_jacobi_init, density1, energy1,
        work_array_1, work_array_2, work_array_3, u, coefficient);
}

void CloverleafCudaChunk::tea_leaf_kernel_jacobi
(double rx, double ry, double* error)
{
    CUDALAUNCH(device_tea_leaf_jacobi_copy_u, u, work_array_4);

    CUDALAUNCH(device_tea_leaf_jacobi_solve, rx, ry, work_array_1, work_array_2,
        work_array_3, u, work_array_4, reduce_buf_1);

    *error = thrust::reduce(reduce_ptr_1, reduce_ptr_1 + num_blocks, 0.0);
}

/********************/

// used by both
extern "C" void tea_leaf_kernel_finalise_cuda_
(void)
{
    cuda_chunk.tea_leaf_finalise();
}

// both
void CloverleafCudaChunk::tea_leaf_finalise
(void)
{
    CUDALAUNCH(device_tea_leaf_finalise, density1, u, energy1);
}

