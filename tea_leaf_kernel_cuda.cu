#include "hip/hip_runtime.h"
#include "cuda_common.hpp"

#include "kernel_files/tea_leaf_common.cuknl"
#include "kernel_files/tea_leaf_jacobi.cuknl"
#include "kernel_files/tea_leaf_cg.cuknl"
#include "kernel_files/tea_leaf_cheby.cuknl"
#include "kernel_files/tea_leaf_ppcg.cuknl"

#include <cassert>

// same as in fortran
#define CONDUCTIVITY 1
#define RECIP_CONDUCTIVITY 2

// copy back dx/dy and calculate rx/ry
void CloverleafCudaChunk::calcrxry
(double dt, double * rx, double * ry)
{
    double dx, dy;

    hipMemcpy(&dx, celldx, sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(&dy, celldy, sizeof(double), hipMemcpyDeviceToHost);

    CUDA_ERR_CHECK;

    *rx = dt/(dx*dx);
    *ry = dt/(dy*dy);
}

/********************/

// Chebyshev solver
extern "C" void tea_leaf_kernel_cheby_copy_u_cuda_
(void)
{
    cuda_chunk.tea_leaf_cheby_copy_u();
}

extern "C" void tea_leaf_calc_2norm_kernel_cuda_
(int* norm_array, double* norm)
{
    cuda_chunk.tea_leaf_calc_2norm_kernel(*norm_array, norm);
}

extern "C" void tea_leaf_kernel_cheby_init_cuda_
(const double * ch_alphas, const double * ch_betas, int* n_coefs,
 const double * rx, const double * ry, const double * theta, double* error)
{
    cuda_chunk.tea_leaf_kernel_cheby_init(ch_alphas, ch_betas, *n_coefs,
        *rx, *ry, *theta, error);
}

extern "C" void tea_leaf_kernel_cheby_iterate_cuda_
(const double * ch_alphas, const double * ch_betas, int *n_coefs,
 const double * rx, const double * ry, const int * cheby_calc_step)
{
    cuda_chunk.tea_leaf_kernel_cheby_iterate(ch_alphas, ch_betas, *n_coefs,
        *rx, *ry, *cheby_calc_step);
}

void CloverleafCudaChunk::tea_leaf_cheby_copy_u
(void)
{
    hipDeviceSynchronize();
    hipMemcpy(u0, u, BUFSZ2D(0, 0), hipMemcpyDeviceToDevice);
}

void CloverleafCudaChunk::tea_leaf_calc_2norm_kernel
(int norm_array, double* norm)
{
    if (norm_array == 0)
    {
        // norm of u0
        CUDALAUNCH(device_tea_leaf_cheby_solve_calc_resid, u0, reduce_buf_1);
    }
    else if (norm_array == 1)
    {
        // norm of r
        CUDALAUNCH(device_tea_leaf_cheby_solve_calc_resid, work_array_2, reduce_buf_1);
    }
    else
    {
        DIE("Invalid value '%d' for norm_array passed, should be [1, 2]", norm_array);
    }

    CUDA_ERR_CHECK;

    *norm = thrust::reduce(reduce_ptr_1, reduce_ptr_1 + num_blocks, 0.0);
}

void CloverleafCudaChunk::upload_ch_coefs
(const double * ch_alphas, const double * ch_betas,
 const int n_coefs)
{
    size_t ch_buf_sz = n_coefs*sizeof(double);

    // upload to device
    hipMalloc((void**) &ch_alphas_device, ch_buf_sz);
    hipMalloc((void**) &ch_betas_device, ch_buf_sz);
    hipMemcpy(ch_alphas_device, ch_alphas, ch_buf_sz, hipMemcpyHostToDevice);
    hipMemcpy(ch_betas_device, ch_betas, ch_buf_sz, hipMemcpyHostToDevice);
}

void CloverleafCudaChunk::tea_leaf_kernel_cheby_init
(const double * ch_alphas, const double * ch_betas, int n_coefs,
 const double rx, const double ry, const double theta, double* error)
{
    assert(tea_solver == TEA_ENUM_CHEBYSHEV);

    upload_ch_coefs(ch_alphas, ch_betas, n_coefs);

    CUDA_ERR_CHECK;

    CUDALAUNCH(device_tea_leaf_cheby_solve_init_p, u, u0,
        work_array_1, work_array_2, work_array_3, work_array_4,
        work_array_5, work_array_6,
        theta, rx, ry, preconditioner_on);

    // update p
    CUDALAUNCH(device_tea_leaf_cheby_solve_calc_u, u, work_array_1);
}

void CloverleafCudaChunk::tea_leaf_kernel_cheby_iterate
(const double * ch_alphas, const double * ch_betas, int n_coefs,
 const double rx, const double ry, const int cheby_calc_step)
{
    CUDALAUNCH(device_tea_leaf_cheby_solve_calc_p, u, u0,
        work_array_1, work_array_2, work_array_3, work_array_4,
        work_array_5, work_array_6,
        ch_alphas_device, ch_betas_device,
        rx, ry, cheby_calc_step-1, preconditioner_on);

    CUDALAUNCH(device_tea_leaf_cheby_solve_calc_u, u, work_array_1);
}

/********************/

// CG solver functions
extern "C" void tea_leaf_kernel_init_cg_cuda_
(const int * coefficient, double * dt, double * rx, double * ry, double * rro)
{
    cuda_chunk.tea_leaf_init_cg(*coefficient, *dt, rx, ry, rro);
}

extern "C" void tea_leaf_kernel_solve_cg_cuda_calc_w_
(const double * rx, const double * ry, double * pw)
{
    cuda_chunk.tea_leaf_kernel_cg_calc_w(*rx, *ry, pw);
}
extern "C" void tea_leaf_kernel_solve_cg_cuda_calc_ur_
(double * alpha, double * rrn)
{
    cuda_chunk.tea_leaf_kernel_cg_calc_ur(*alpha, rrn);
}
extern "C" void tea_leaf_kernel_solve_cg_cuda_calc_p_
(double * beta)
{
    cuda_chunk.tea_leaf_kernel_cg_calc_p(*beta);
}

/********************/

void CloverleafCudaChunk::tea_leaf_init_cg
(int coefficient, double dt, double * rx, double * ry, double * rro)
{
    if (coefficient != CONDUCTIVITY && coefficient != RECIP_CONDUCTIVITY)
    {
        DIE("Unknown coefficient %d passed to tea leaf\n", coefficient);
    }

    assert(tea_solver == TEA_ENUM_CG || tea_solver == TEA_ENUM_CHEBYSHEV || tea_solver == TEA_ENUM_PPCG);

    calcrxry(dt, rx, ry);

    CUDALAUNCH(device_tea_leaf_cg_init_u, density, energy1, u,
        work_array_1, work_array_2, work_array_3, coefficient);

    // init Kx, Ky
    CUDALAUNCH(device_tea_leaf_cg_init_directions, work_array_3, work_array_5, work_array_6);

    // premultiply Kx/Ky
    CUDALAUNCH(device_tea_leaf_init_diag, work_array_5, work_array_6, *rx, *ry);

    // get initial guess in w, r, etc
    CUDALAUNCH(device_tea_leaf_cg_init_others, reduce_buf_2, u,
        work_array_1, work_array_2, work_array_3, work_array_4, z,
        work_array_5, work_array_6, *rx, *ry, preconditioner_on);

    *rro = thrust::reduce(reduce_ptr_2, reduce_ptr_2 + num_blocks, 0.0);
}

void CloverleafCudaChunk::tea_leaf_kernel_cg_calc_w
(double rx, double ry, double* pw)
{
    CUDALAUNCH(device_tea_leaf_cg_solve_calc_w, reduce_buf_3,
        work_array_1, work_array_4, work_array_5, work_array_6, rx, ry);

    *pw = thrust::reduce(reduce_ptr_3, reduce_ptr_3 + num_blocks, 0.0);
}

void CloverleafCudaChunk::tea_leaf_kernel_cg_calc_ur
(double alpha, double* rrn)
{
    CUDALAUNCH(device_tea_leaf_cg_solve_calc_ur, alpha, reduce_buf_4, u, work_array_1,
        work_array_2, work_array_4, z, work_array_3, preconditioner_on);

    *rrn = thrust::reduce(reduce_ptr_4, reduce_ptr_4 + num_blocks, 0.0);
}

void CloverleafCudaChunk::tea_leaf_kernel_cg_calc_p
(double beta)
{
    CUDALAUNCH(device_tea_leaf_cg_solve_calc_p, beta, work_array_1, work_array_2, z,
        preconditioner_on);
}

/********************/

// jacobi solver functions
extern "C" void tea_leaf_kernel_init_cuda_
(const int * coefficient, double * dt, double * rx, double * ry)
{
    cuda_chunk.tea_leaf_init_jacobi(*coefficient, *dt, rx, ry);
}

extern "C" void tea_leaf_kernel_solve_cuda_
(const double * rx, const double * ry, double * error)
{
    cuda_chunk.tea_leaf_kernel_jacobi(*rx, *ry, error);
}

// jacobi
void CloverleafCudaChunk::tea_leaf_init_jacobi
(int coefficient, double dt, double * rx, double * ry)
{
    if (coefficient != CONDUCTIVITY && coefficient != RECIP_CONDUCTIVITY)
    {
        DIE("Unknown coefficient %d passed to tea leaf\n", coefficient);
    }

    calcrxry(dt, rx, ry);

    CUDALAUNCH(device_tea_leaf_jacobi_init, density, energy1,
        work_array_5, work_array_6, work_array_3, u, coefficient);
}

void CloverleafCudaChunk::tea_leaf_kernel_jacobi
(double rx, double ry, double* error)
{
    CUDALAUNCH(device_tea_leaf_jacobi_copy_u, u, work_array_4);

    CUDALAUNCH(device_tea_leaf_jacobi_solve, rx, ry, work_array_5, work_array_6,
        work_array_3, u, work_array_4, reduce_buf_1);

    *error = *thrust::max_element(reduce_ptr_1, reduce_ptr_1 + num_blocks);
}

/********************/

// used by both
extern "C" void tea_leaf_kernel_finalise_cuda_
(void)
{
    cuda_chunk.tea_leaf_finalise();
}

extern "C" void tea_leaf_calc_residual_cuda_
(void)
{
    cuda_chunk.tea_leaf_calc_residual();
}

// both
void CloverleafCudaChunk::tea_leaf_finalise
(void)
{
    CUDALAUNCH(device_tea_leaf_finalise, density, u, energy1);
}

void CloverleafCudaChunk::tea_leaf_calc_residual
(void)
{
    CUDALAUNCH(device_tea_leaf_calc_residual, u, u0, work_array_3,
        work_array_5, work_array_6);
}

/********************/

extern "C" void tea_leaf_kernel_ppcg_init_cuda_
(const double * ch_alphas, const double * ch_betas,
 double* theta, int* n_inner_steps)
{
    cuda_chunk.ppcg_init(ch_alphas, ch_betas, *theta, *n_inner_steps);
}

extern "C" void tea_leaf_kernel_ppcg_init_p_cuda_
(double * rro)
{
    cuda_chunk.ppcg_init_p(rro);
}

extern "C" void tea_leaf_kernel_ppcg_init_sd_cuda_
(const double * theta)
{
    cuda_chunk.ppcg_init_sd(*theta);
}

extern "C" void tea_leaf_kernel_ppcg_inner_cuda_
(int * ppcg_cur_step)
{
    cuda_chunk.ppcg_inner(*ppcg_cur_step);
}

void CloverleafCudaChunk::ppcg_init
(const double * ch_alphas, const double * ch_betas,
 const double theta, const int n_inner_steps)
{
    if(preconditioner_on)
    {
        DIE("Preconditioner does not work with PPCG solver - disable in input file");
    }

    upload_ch_coefs(ch_alphas, ch_betas, n_inner_steps);
}

void CloverleafCudaChunk::ppcg_init_p
(double * rro)
{
    // FIXME work_arrays - rename to u, p, r, etc
    CUDALAUNCH(device_tea_leaf_ppcg_solve_init_p, work_array_1,
        work_array_3, work_array_4, reduce_buf_1);

    *rro = thrust::reduce(reduce_ptr_1, reduce_ptr_1 + num_blocks, 0.0);
}

void CloverleafCudaChunk::ppcg_init_sd
(double theta)
{
    CUDALAUNCH(device_tea_leaf_ppcg_solve_init_sd, work_array_3,
        work_array_4, work_array_8, theta);
}

void CloverleafCudaChunk::ppcg_inner
(int ppcg_cur_step)
{
    CUDALAUNCH(device_tea_leaf_ppcg_solve_update_r, u, work_array_3,
        work_array_5, work_array_6, work_array_8);

    CUDALAUNCH(device_tea_leaf_ppcg_solve_calc_sd, work_array_3,
        work_array_4, work_array_8, ch_alphas_device, ch_betas_device,
        ppcg_cur_step - 1);
}


