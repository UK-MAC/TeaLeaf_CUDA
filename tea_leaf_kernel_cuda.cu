#include "hip/hip_runtime.h"
#include "cuda_common.hpp"

// same as in fortran
#define COEF_CONDUCTIVITY 1
#define COEF_RECIP_CONDUCTIVITY 2

#include "kernel_files/tea_leaf_common.cuknl"
#include "kernel_files/tea_leaf_jacobi.cuknl"
#include "kernel_files/tea_leaf_cg.cuknl"
#include "kernel_files/tea_leaf_cheby.cuknl"
#include "kernel_files/tea_leaf_ppcg.cuknl"

#include <cassert>

// copy back dx/dy and calculate rx/ry
void CloverleafCudaChunk::calcrxry
(double dt, double * rx, double * ry)
{
    double dx, dy;

    hipMemcpy(&dx, halo_exchange_depth + celldx, sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(&dy, halo_exchange_depth + celldy, sizeof(double), hipMemcpyDeviceToHost);

    hipDeviceSynchronize();

    CUDA_ERR_CHECK;

    *rx = dt/(dx*dx);
    *ry = dt/(dy*dy);
}

extern "C" void tea_leaf_calc_2norm_kernel_cuda_
(int* norm_array, double* norm)
{
    cuda_chunk.tea_leaf_calc_2norm_kernel(*norm_array, norm);
}

/********************/

extern "C" void tea_leaf_cheby_init_kernel_cuda_
(const double * ch_alphas, const double * ch_betas, int* n_coefs,
 const double * theta)
{
    cuda_chunk.tea_leaf_kernel_cheby_init(ch_alphas, ch_betas, *n_coefs,
        *theta);
}

extern "C" void tea_leaf_cheby_iterate_kernel_cuda_
(const int * cheby_calc_step)
{
    cuda_chunk.tea_leaf_kernel_cheby_iterate(*cheby_calc_step);
}

void CloverleafCudaChunk::tea_leaf_calc_2norm_kernel
(int norm_array, double* norm)
{
    if (norm_array == 0)
    {
        // norm of u0
        CUDALAUNCH(device_tea_leaf_calc_2norm, u0, u0, reduce_buf_1);
    }
    else if (norm_array == 1)
    {
        // norm of r
        CUDALAUNCH(device_tea_leaf_calc_2norm, vector_r, vector_r, reduce_buf_1);
    }
    else if (norm_array == 2)
    {
        CUDALAUNCH(device_tea_leaf_calc_2norm, vector_r, vector_z, reduce_buf_1);
    }
    else
    {
        DIE("Invalid value '%d' for norm_array passed, should be [0, 1, 2]", norm_array);
    }

    CUDA_ERR_CHECK;

    *norm = thrust::reduce(reduce_ptr_1, reduce_ptr_1 + num_blocks, 0.0);
}

void CloverleafCudaChunk::upload_ch_coefs
(const double * ch_alphas, const double * ch_betas,
 const int n_coefs)
{
    size_t ch_buf_sz = n_coefs*sizeof(double);

    if (ch_alphas_device == NULL && ch_betas_device == NULL)
    {
        hipMalloc((void**) &ch_alphas_device, ch_buf_sz);
        hipMalloc((void**) &ch_betas_device, ch_buf_sz);
    }

    // upload to device
    hipMemcpy(ch_alphas_device, ch_alphas, ch_buf_sz, hipMemcpyHostToDevice);
    hipMemcpy(ch_betas_device, ch_betas, ch_buf_sz, hipMemcpyHostToDevice);
}

void CloverleafCudaChunk::tea_leaf_kernel_cheby_init
(const double * ch_alphas, const double * ch_betas, int n_coefs,
 const double theta)
{
    assert(tea_solver == TEA_ENUM_CHEBYSHEV);

    upload_ch_coefs(ch_alphas, ch_betas, n_coefs);

    CUDA_ERR_CHECK;

    CUDALAUNCH(device_tea_leaf_cheby_solve_init_p, u, u0,
        vector_p, vector_r, vector_w, tri_cp, tri_bfp,
        vector_Mi, vector_Kx, vector_Ky,
        theta);

    // update p
    CUDALAUNCH(device_tea_leaf_cheby_solve_calc_u, u, vector_p);
}

void CloverleafCudaChunk::tea_leaf_kernel_cheby_iterate
(const int cheby_calc_step)
{
    CUDALAUNCH(device_tea_leaf_cheby_solve_calc_p, u, u0,
        vector_p, vector_r, vector_w, tri_cp, tri_bfp,
        vector_Mi, vector_Kx, vector_Ky,
        ch_alphas_device, ch_betas_device,
        cheby_calc_step-1);

    CUDALAUNCH(device_tea_leaf_cheby_solve_calc_u, u, vector_p);
}

/********************/

// CG solver functions
extern "C" void tea_leaf_cg_init_kernel_cuda_
(double * rro)
{
    cuda_chunk.tea_leaf_init_cg(rro);
}

extern "C" void tea_leaf_cg_calc_w_kernel_cuda_
(double * pw)
{
    cuda_chunk.tea_leaf_kernel_cg_calc_w(pw);
}
extern "C" void tea_leaf_cg_calc_ur_kernel_cuda_
(double * alpha, double * rrn)
{
    cuda_chunk.tea_leaf_kernel_cg_calc_ur(*alpha, rrn);
}
extern "C" void tea_leaf_cg_calc_p_kernel_cuda_
(double * beta)
{
    cuda_chunk.tea_leaf_kernel_cg_calc_p(*beta);
}

/********************/

void CloverleafCudaChunk::tea_leaf_init_cg
(double * rro)
{
    assert(tea_solver == TEA_ENUM_CG || tea_solver == TEA_ENUM_CHEBYSHEV || tea_solver == TEA_ENUM_PPCG);

    if (preconditioner_type == TL_PREC_JAC_BLOCK)
    {
        // TODO preconditioners
    }
    else if (preconditioner_type == TL_PREC_JAC_DIAG)
    {
        CUDALAUNCH(device_tea_leaf_init_jac_diag, vector_Mi, vector_Kx, vector_Ky);
    }

    // init Kx, Ky
    CUDALAUNCH(device_tea_leaf_cg_solve_init_p, vector_p, vector_r,
        vector_z, vector_Mi, reduce_buf_2);

    *rro = thrust::reduce(reduce_ptr_2, reduce_ptr_2 + num_blocks, 0.0);
}

void CloverleafCudaChunk::tea_leaf_kernel_cg_calc_w
(double* pw)
{
    CUDALAUNCH(device_tea_leaf_cg_solve_calc_w, reduce_buf_3,
        vector_p, vector_w, vector_Kx, vector_Ky);

    *pw = thrust::reduce(reduce_ptr_3, reduce_ptr_3 + num_blocks, 0.0);
}

void CloverleafCudaChunk::tea_leaf_kernel_cg_calc_ur
(double alpha, double* rrn)
{
    CUDALAUNCH(device_tea_leaf_cg_solve_calc_ur, alpha, u, vector_p,
        vector_r, vector_w, vector_z, tri_cp, tri_bfp,
        vector_Mi, vector_Kx, vector_Ky, reduce_buf_4);

    *rrn = thrust::reduce(reduce_ptr_4, reduce_ptr_4 + num_blocks, 0.0);
}

void CloverleafCudaChunk::tea_leaf_kernel_cg_calc_p
(double beta)
{
    CUDALAUNCH(device_tea_leaf_cg_solve_calc_p, beta, vector_p, vector_r, vector_z);
}

/********************/

extern "C" void tea_leaf_jacobi_solve_kernel_cuda_
(double * error)
{
    cuda_chunk.tea_leaf_kernel_jacobi(error);
}

void CloverleafCudaChunk::tea_leaf_kernel_jacobi
(double* error)
{
    CUDALAUNCH(device_tea_leaf_jacobi_copy_u, u, vector_Mi);

    CUDALAUNCH(device_tea_leaf_jacobi_solve, vector_Kx, vector_Ky,
        vector_w, u, vector_Mi, reduce_buf_1);

    *error = *thrust::max_element(reduce_ptr_1, reduce_ptr_1 + num_blocks);
}

/********************/

extern "C" void tea_leaf_common_init_kernel_cuda_
(const int * coefficient, double * dt, double * rx, double * ry,
 int * chunk_neighbours, int * zero_boundary, int * reflective_boundary)
{
    cuda_chunk.tea_leaf_common_init(*coefficient, *dt, rx, ry,
        chunk_neighbours, zero_boundary, *reflective_boundary);
}

// used by both
extern "C" void tea_leaf_common_finalise_kernel_cuda_
(void)
{
    cuda_chunk.tea_leaf_finalise();
}

extern "C" void tea_leaf_calc_residual_cuda_
(void)
{
    cuda_chunk.tea_leaf_calc_residual();
}

void CloverleafCudaChunk::tea_leaf_common_init
(int coefficient, double dt, double * rx, double * ry,
 int * chunk_neighbours, int * zero_boundary, int reflective_boundary)
{
    if (coefficient != COEF_CONDUCTIVITY && coefficient != COEF_RECIP_CONDUCTIVITY)
    {
        DIE("Unknown coefficient %d passed to tea leaf\n", coefficient);
    }

    calcrxry(dt, rx, ry);

    CUDALAUNCH(device_tea_leaf_init_common, density, energy1,
        vector_Kx, vector_Ky, *rx, *ry, coefficient);

    if (!reflective_boundary)
    {
        int zero_left = chunk_neighbours[CHUNK_left - 1] && zero_boundary[CHUNK_left - 1];
        int zero_right = chunk_neighbours[CHUNK_right - 1] && zero_boundary[CHUNK_right - 1];
        int zero_bottom = chunk_neighbours[CHUNK_bottom - 1] && zero_boundary[CHUNK_bottom - 1];
        int zero_top = chunk_neighbours[CHUNK_top - 1] && zero_boundary[CHUNK_top - 1];

        CUDALAUNCH(device_tea_leaf_zero_boundaries, vector_Kx, vector_Ky,
            zero_left,
            zero_right,
            zero_bottom,
            zero_top);
    }

    generate_chunk_init_u(energy1);
}

// both
void CloverleafCudaChunk::tea_leaf_finalise
(void)
{
    CUDALAUNCH(device_tea_leaf_finalise, density, u, energy1);
}

void CloverleafCudaChunk::tea_leaf_calc_residual
(void)
{
    CUDALAUNCH(device_tea_leaf_calc_residual, u, u0, vector_r,
        vector_Kx, vector_Ky);
}

/********************/

extern "C" void tea_leaf_ppcg_init_cuda_
(const double * ch_alphas, const double * ch_betas,
 int* n_inner_steps)
{
    cuda_chunk.ppcg_init(ch_alphas, ch_betas, *n_inner_steps);
}

extern "C" void tea_leaf_ppcg_init_sd_kernel_cuda_
(const double * theta)
{
    cuda_chunk.ppcg_init_sd(*theta);
}

extern "C" void tea_leaf_ppcg_inner_kernel_cuda_
(int * ppcg_cur_step, int * bounds_extra,
 int * chunk_neighbours)
{
    cuda_chunk.ppcg_inner(*ppcg_cur_step, *bounds_extra, chunk_neighbours);
}

void CloverleafCudaChunk::ppcg_init
(const double * ch_alphas, const double * ch_betas,
 const int n_inner_steps)
{
    upload_ch_coefs(ch_alphas, ch_betas, n_inner_steps);
}

void CloverleafCudaChunk::ppcg_init_sd
(double theta)
{
    CUDALAUNCH(device_tea_leaf_ppcg_solve_init_sd, vector_r,
        vector_Mi, vector_sd, theta);
}

void CloverleafCudaChunk::ppcg_inner
(int ppcg_cur_step, int bounds_extra,
 int * chunk_neighbours)
{
    // TODO offsets
    CUDALAUNCH(device_tea_leaf_ppcg_solve_update_r, u, vector_r,
        vector_Kx, vector_Ky, vector_sd);

    CUDALAUNCH(device_tea_leaf_ppcg_solve_calc_sd, vector_r,
        vector_Mi, vector_sd, ch_alphas_device, ch_betas_device,
        ppcg_cur_step - 1);
}

