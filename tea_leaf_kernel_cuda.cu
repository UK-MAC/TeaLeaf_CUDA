#include "hip/hip_runtime.h"
#include "cuda_common.hpp"

#include "kernel_files/tea_leaf_common.cuknl"
#include "kernel_files/tea_leaf_jacobi.cuknl"
#include "kernel_files/tea_leaf_cg.cuknl"
#include "kernel_files/tea_leaf_cheby.cuknl"
#include "kernel_files/tea_leaf_ppcg.cuknl"

#include <cassert>

// same as in fortran
#define CONDUCTIVITY 1
#define RECIP_CONDUCTIVITY 2

// copy back dx/dy and calculate rx/ry
void CloverleafCudaChunk::calcrxry
(double dt, double * rx, double * ry)
{
    double dx, dy;

    hipMemcpy(&dx, celldx, sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(&dy, celldy, sizeof(double), hipMemcpyDeviceToHost);

    CUDA_ERR_CHECK;

    *rx = dt/(dx*dx);
    *ry = dt/(dy*dy);
}

extern "C" void tea_leaf_calc_2norm_kernel_cuda_
(int* norm_array, double* norm)
{
    cuda_chunk.tea_leaf_calc_2norm_kernel(*norm_array, norm);
}

/********************/

extern "C" void tea_leaf_cheby_init_kernel_cuda_
(const double * ch_alphas, const double * ch_betas, int* n_coefs,
 const double * rx, const double * ry, const double * theta)
{
    cuda_chunk.tea_leaf_kernel_cheby_init(ch_alphas, ch_betas, *n_coefs,
        *rx, *ry, *theta);
}

extern "C" void tea_leaf_cheby_iterate_kernel_cuda_
(const double * rx, const double * ry, const int * cheby_calc_step)
{
    cuda_chunk.tea_leaf_kernel_cheby_iterate(*rx, *ry, *cheby_calc_step);
}

void CloverleafCudaChunk::tea_leaf_calc_2norm_kernel
(int norm_array, double* norm)
{
    if (norm_array == 0)
    {
        // norm of u0
        CUDALAUNCH(device_tea_leaf_common_calc_2norm, u0, u0, reduce_buf_1);
    }
    else if (norm_array == 1)
    {
        // norm of r
        CUDALAUNCH(device_tea_leaf_common_calc_2norm, vector_r, vector_r, reduce_buf_1);
    }
    else if (norm_array == 2)
    {
        CUDALAUNCH(device_tea_leaf_common_calc_2norm, vector_r, vector_z, reduce_buf_1);
    }
    else
    // TODO
    {
        DIE("Invalid value '%d' for norm_array passed, should be [1, 2]", norm_array);
    }

    CUDA_ERR_CHECK;

    *norm = thrust::reduce(reduce_ptr_1, reduce_ptr_1 + num_blocks, 0.0);
}

void CloverleafCudaChunk::upload_ch_coefs
(const double * ch_alphas, const double * ch_betas,
 const int n_coefs)
{
    size_t ch_buf_sz = n_coefs*sizeof(double);

    // upload to device
    hipMalloc((void**) &ch_alphas_device, ch_buf_sz);
    hipMalloc((void**) &ch_betas_device, ch_buf_sz);
    hipMemcpy(ch_alphas_device, ch_alphas, ch_buf_sz, hipMemcpyHostToDevice);
    hipMemcpy(ch_betas_device, ch_betas, ch_buf_sz, hipMemcpyHostToDevice);
}

void CloverleafCudaChunk::tea_leaf_kernel_cheby_init
(const double * ch_alphas, const double * ch_betas, int n_coefs,
 const double rx, const double ry, const double theta)
{
    assert(tea_solver == TEA_ENUM_CHEBYSHEV);

    upload_ch_coefs(ch_alphas, ch_betas, n_coefs);

    CUDA_ERR_CHECK;

    CUDALAUNCH(device_tea_leaf_cheby_solve_init_p, u, u0,
        vector_p, vector_r, vector_w, vector_Mi,
        vector_Kx, vector_Ky,
        theta, rx, ry, preconditioner_on);

    // update p
    CUDALAUNCH(device_tea_leaf_cheby_solve_calc_u, u, vector_p);
}

void CloverleafCudaChunk::tea_leaf_kernel_cheby_iterate
(const double rx, const double ry, const int cheby_calc_step)
{
    CUDALAUNCH(device_tea_leaf_cheby_solve_calc_p, u, u0,
        vector_p, vector_r, vector_w, vector_Mi,
        vector_Kx, vector_Ky,
        ch_alphas_device, ch_betas_device,
        rx, ry, cheby_calc_step-1, preconditioner_on);

    CUDALAUNCH(device_tea_leaf_cheby_solve_calc_u, u, vector_p);
}

/********************/

// CG solver functions
extern "C" void tea_leaf_cg_init_kernel_cuda_
(double * rro)
{
    cuda_chunk.tea_leaf_init_cg(rro);
}

extern "C" void tea_leaf_cg_calc_w_kernel_cuda_
(double * pw)
{
    cuda_chunk.tea_leaf_kernel_cg_calc_w(pw);
}
extern "C" void tea_leaf_cg_calc_ur_kernel_cuda_
(double * alpha, double * rrn)
{
    cuda_chunk.tea_leaf_kernel_cg_calc_ur(*alpha, rrn);
}
extern "C" void tea_leaf_cg_calc_p_kernel_cuda_
(double * beta)
{
    cuda_chunk.tea_leaf_kernel_cg_calc_p(*beta);
}

/********************/

void CloverleafCudaChunk::tea_leaf_init_cg
(int coefficient, double dt, double * rx, double * ry, double * rro)
{
    assert(tea_solver == TEA_ENUM_CG || tea_solver == TEA_ENUM_CHEBYSHEV || tea_solver == TEA_ENUM_PPCG);

    // TODO preconditioners

    // init Kx, Ky
    CUDALAUNCH(device_tea_leaf_cg_init_p, vector_w, vector_Kx, vector_Ky);

    *rro = thrust::reduce(reduce_ptr_2, reduce_ptr_2 + num_blocks, 0.0);
}

void CloverleafCudaChunk::tea_leaf_kernel_cg_calc_w
(double rx, double ry, double* pw)
{
    CUDALAUNCH(device_tea_leaf_cg_solve_calc_w, reduce_buf_3,
        vector_p, vector_Mi, vector_Kx, vector_Ky, rx, ry);

    *pw = thrust::reduce(reduce_ptr_3, reduce_ptr_3 + num_blocks, 0.0);
}

void CloverleafCudaChunk::tea_leaf_kernel_cg_calc_ur
(double alpha, double* rrn)
{
    CUDALAUNCH(device_tea_leaf_cg_solve_calc_ur, alpha, reduce_buf_4, u, vector_p,
        vector_r, vector_Mi, z, vector_w, preconditioner_on);

    *rrn = thrust::reduce(reduce_ptr_4, reduce_ptr_4 + num_blocks, 0.0);
}

void CloverleafCudaChunk::tea_leaf_kernel_cg_calc_p
(double beta)
{
    CUDALAUNCH(device_tea_leaf_cg_solve_calc_p, beta, vector_p, vector_r, z,
        preconditioner_on);
}

/********************/

extern "C" void tea_leaf_jacobi_solve_kernel_cuda_
(double * error)
{
    cuda_chunk.tea_leaf_kernel_jacobi(error);
}

void CloverleafCudaChunk::tea_leaf_kernel_jacobi
(double* error)
{
    CUDALAUNCH(device_tea_leaf_jacobi_copy_u, u, vector_Mi);

    CUDALAUNCH(device_tea_leaf_jacobi_solve, rx, ry, vector_Kx, vector_Ky,
        vector_w, u, vector_Mi, reduce_buf_1);

    *error = *thrust::max_element(reduce_ptr_1, reduce_ptr_1 + num_blocks);
}

/********************/

extern "C" void tea_leaf_common_init_kernel_cuda_
(const int * coefficient, double * dt, double * rx, double * ry,
 int * chunk_neighbours, int * zero_boundary, int * reflective_boundary)
{
    cuda_chunk.tea_leaf_common_init(*coefficient, *dt, rx, ry,
        chunk_neighbours, zero_boundary, *reflective_boundary);
}

// used by both
extern "C" void tea_leaf_common_finalise_kernel_cuda_
(void)
{
    cuda_chunk.tea_leaf_finalise();
}

extern "C" void tea_leaf_calc_residual_cuda_
(void)
{
    cuda_chunk.tea_leaf_calc_residual();
}

void CloverleafCudaChunk::tea_leaf_init_jacobi
(int coefficient, double dt, double * rx, double * ry)
{
    if (coefficient != CONDUCTIVITY && coefficient != RECIP_CONDUCTIVITY)
    {
        DIE("Unknown coefficient %d passed to tea leaf\n", coefficient);
    }

    calcrxry(dt, rx, ry);

    CUDALAUNCH(device_tea_leaf_jacobi_init, density, energy1,
        vector_Kx, vector_Ky, vector_w, u, coefficient);
}

// both
void CloverleafCudaChunk::tea_leaf_finalise
(void)
{
    CUDALAUNCH(device_tea_leaf_finalise, density, u, energy1);
}

void CloverleafCudaChunk::tea_leaf_calc_residual
(void)
{
    CUDALAUNCH(device_tea_leaf_calc_residual, u, u0, vector_w,
        vector_Kx, vector_Ky);
}

/********************/

extern "C" void tea_leaf_kernel_ppcg_init_cuda_
(const double * ch_alphas, const double * ch_betas,
 int* n_inner_steps)
{
    cuda_chunk.ppcg_init(ch_alphas, ch_betas, *n_inner_steps);
}

extern "C" void tea_leaf_ppcg_init_sd_kernel_cuda_
(const double * theta)
{
    cuda_chunk.ppcg_init_sd(*theta);
}

extern "C" void tea_leaf_ppcg_inner_kernel_cuda
(int * ppcg_cur_step)
{
    cuda_chunk.ppcg_inner(*ppcg_cur_step);
}

void CloverleafCudaChunk::ppcg_init
(const double * ch_alphas, const double * ch_betas,
 const int n_inner_steps)
{
    upload_ch_coefs(ch_alphas, ch_betas, n_inner_steps);
}

void CloverleafCudaChunk::ppcg_init_sd
(double theta)
{
    CUDALAUNCH(device_tea_leaf_ppcg_solve_init_sd, vector_r,
        vector_Mi, vector_sd, theta);
}

void CloverleafCudaChunk::ppcg_inner
(int ppcg_cur_step)
{
    // TODO offsets
    CUDALAUNCH(device_tea_leaf_ppcg_solve_update_r, u, vector_r,
        vector_Kx, vector_Ky, vector_sd);

    CUDALAUNCH(device_tea_leaf_ppcg_solve_calc_sd, vector_r,
        vector_Mi, vector_sd, ch_alphas_device, ch_betas_device,
        ppcg_cur_step - 1);
}

